#include "hip/hip_runtime.h"
﻿#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu\optixu_matrix_namespace.h>
#include "cu_common.h"
#include "../../host_device/data_renderer.h"
#include "../../host_device/data_light.h";
#include "../../host_device/data_instance.h"
#include "intersection.h"

using namespace optix;

#include "light_model_data.h"

RT_FUNC_F float DistanceSquared(const Pos& p1,const Pos& p2){
  const Dir tmp(p1-p2);
  return dot(tmp,tmp);
}

/*
RT_FUNC_F float4 Multiply(const Matrix4x4& m,const Dir& d){
	return m.getCol(0)*d.x+m.getCol(1)*d.y+m.getCol(2)*d.z;
}
rtDeclareVariable(SData_Renderer,V_C_DATA_RENDERER,,);
*/


#pragma region MONTECARLO
/**
* v1 and v2 must be at same space with n;
*/
RT_FUNC_F bool SameHemisphere(const Dir& v1,const Dir& v2,const Nor& n){
	return dot(v1,n)*dot(v2,n)>0.f;
}

RT_FUNC_S Dir UniformSampleHemisphere(float u1,float u2){
	float z=u1;
	float r=sqrtf(max(0.f,1.f-z*z));
	float phi=M_2PIf * u2;
	float x=r * cosf(phi);
	float y=r * sinf(phi);
	return make_float3(x,y,z);
}

RT_FUNC_S Dir UniformSampleSphere(float u1,float u2){
	float z=1.f-2.f * u1;
	float r=sqrtf(max(0.f,1.f-z*z));
	float phi=M_2PIf * u2;
	float x=r * cosf(phi);
	float y=r * sinf(phi);
	return make_float3(x,y,z);
}

RT_FUNC_F float UniformSpherePdf(){
	return 1.f/M_4PIf;
}

RT_CALLABLE_PROGRAM void CF_UNIFORMSAMPLETRIANGLE(RandomFloat1 u1,RandomFloat1 u2,RandomVariable1& u,RandomVariable1& v){
	float su1=sqrtf(u1);
	u=1.f-su1;
	v=u2 * su1;
}

RT_CALLABLE_PROGRAM void CF_CONCENTRICSAMPLEDISK(RandomFloat1 u1,RandomFloat1 u2,RandomVariable1* dx,RandomVariable1* dy){
	float r,theta;
	// Map uniform random numbers to $[-1,1]^2$
	float sx=2.f*u1-1.f;
	float sy=2.f*u2-1.f;

	// Map square to $(r,\theta)$

	// Handle degeneracy at the origin
	if(sx==0.0f && sy==0.0f){
		*dx=0.0f;
		*dy=0.0f;
		return;
	}
	if(sx>=-sy){
		if(sx>sy){
			// Handle first region of disk
			r=sx;
			if(sy>0.0f) theta=sy/r;
			else theta=8.0f+sy/r;
		} else{
			// Handle second region of disk
			r=sy;
			theta=2.0f-sx/r;
		}
	} else{
		if(sx<=sy){
			// Handle third region of disk
			r=-sx;
			theta=4.0f-sy/r;
		} else{
			// Handle fourth region of disk
			r=-sy;
			theta=6.0f+sx/r;
		}
	}
	theta*=M_PIf*.25f;
	*dx=r * cosf(theta);
	*dy=r * sinf(theta);
}

/**
* returns a vector in hemisphere local space, which has z as surface normal;
* 采样具有cos半球分部；
* 均匀的圆盘采样分布点映射到半球面后会使其更加集中于法线方向；
*/
RT_FUNC_S Dir CosineSampleHemisphere(RandomFloat1 u1,RandomFloat1 u2){
	Dir ret;
	ConcentricSampleDisk(u1,u2,&ret.x,&ret.y);
	ret.z=sqrtf(fmaxf(0.f,1.f-ret.x*ret.x-ret.y*ret.y));
	return ret;
}

RT_FUNC_S Dir CosineSampleHemisphere_hack(RandomFloat1 u1,RandomFloat1 u2){
	const float phi=M_2PIf * u2;
	Dir p;
	p.x=u1 * cosf(phi);
	p.y=u1 * sinf(phi);

	// Project up to hemisphere.
	p.z=sqrtf(fmaxf(0.0f,1.0f-p.x*p.x-p.y*p.y));
	return p;
}

RT_CALLABLE_PROGRAM Dir CF_COSINESAMPLEHEMISPHERE_QUICKER(const RandomFloat2& u,float quadrant,float max=4.f){
	// Uniformly sample disk.
	const float r=sqrtf(u.x);
	const float phi=(quadrant+u.y)*M_2PIf/max;
	Dir p;
	p.x=r * cosf(phi);
	p.y=r * sinf(phi);

	// Project up to hemisphere.
	p.z=sqrtf(fmaxf(0.0f,1.0f-p.x*p.x-p.y*p.y));
	return p;
}

RT_CALLABLE_PROGRAM float CF_CHECKERBOARD3(float3 hitPos,float size){
	hitPos+=make_float3(0.001f); // small epsilon so planes don't coincide with scene geometry
	int3 c;

	c.x=abs((int)floor((hitPos.x/size)));
	c.y=abs((int)floor((hitPos.y/size)));
	c.z=abs((int)floor((hitPos.z/size)));

	if((c.x%2)^(c.y%2)^(c.z%2)) return 1.0f;
	return 0.0f;
}

/**
* d1,d2 must be normalized
*/

// 右手坐标系  Z 朝上
RT_FUNC_F Nor SphericalDirection(float sintheta,float costheta,float phi){
	return make_float3(sintheta * cosf(phi),sintheta * sinf(phi),costheta);
}

RT_FUNC_F float CosTheta(const Nor& v,const Nor& n){
	return dot(v,n);
}
RT_FUNC_F float Cos2Theta(const Nor& v,const Nor& n){
	float f=dot(v,n);
	return f*f;
}
RT_FUNC_F float Sin2Theta(const Nor& v,const Nor& n){
	return fmaxf(0.f,1.f-Cos2Theta(v,n));
}
RT_FUNC_F float SinTheta(const Nor& v,const Nor& n){
	return sqrtf(Sin2Theta(v,n));
}

RT_FUNC_F float TanTheta(const Nor& v,const Nor& n){
	return SinTheta(v,n)/CosTheta(v,n);
}
RT_FUNC_F float Tan2Theta(const Nor& v,const Nor& n){
	return Sin2Theta(v,n)/Cos2Theta(v,n);
}
RT_FUNC_F float CosPhi(const Nor& v,const Nor& n,const Nor& t){
	float f=SinTheta(v,n);
	return f==0.f?1.f:clamp(dot(n,t)/f,-1.f,1.f);
}
RT_FUNC_F float SinPhi(const Nor& v,const Nor& n,const Nor& b){
	float f=SinTheta(v,n);
	return f==0.f?0.f:clamp(dot(n,b)/f,-1.f,1.f);
}
RT_FUNC_F float Cos2Phi(const Nor& v,const Nor& n,const Nor& t){
	return CosPhi(v,n,t)*CosPhi(v,n,t);
}
RT_FUNC_F float Sin2Phi(const Nor& v,const Nor& n,const Nor& b){
	return SinPhi(v,n,b)*SinPhi(v,n,b);
}

RT_FUNC_S float ErfInv(float x){
	float w,p;
	x=clamp(x,-.99999f,.99999f);
	w=-logf((1.f-x) * (1.f+x));
	if(w<5){
		w=w-2.5f;
		p=2.81022636e-08f;
		p=3.43273939e-07f+p * w;
		p=-3.5233877e-06f+p * w;
		p=-4.39150654e-06f+p * w;
		p=0.00021858087f+p * w;
		p=-0.00125372503f+p * w;
		p=-0.00417768164f+p * w;
		p=0.246640727f+p * w;
		p=1.50140941f+p * w;
	} else{
		w=sqrtf(w)-3.f;
		p=-0.000200214257f;
		p=0.000100950558f+p * w;
		p=0.00134934322f+p * w;
		p=-0.00367342844f+p * w;
		p=0.00573950773f+p * w;
		p=-0.0076224613f+p * w;
		p=0.00943887047f+p * w;
		p=1.00167406f+p * w;
		p=2.83297682f+p * w;
	}
	return p * x;
}

RT_FUNC_S float Erf(float x){
	// constants
	float a1=0.254829592f;
	float a2=-0.284496736f;
	float a3=1.421413741f;
	float a4=-1.453152027f;
	float a5=1.061405429f;
	float p=0.3275911f;

	// Save the sign of x
	int sign=1;
	if(x<0) sign=-1;
	x=abs(x);

	// A&S formula 7.1.26
	float t=1.f/(1.f+p * x);
	float y=1.f-(((((a5 * t+a4) * t)+a3) * t+a2) * t+a1) * t * expf(-x * x);

	return sign * y;
}


#pragma endregion

#pragma region RANDOM
namespace{

template<unsigned int N>
static __device__ __inline__ unsigned int tea(unsigned int val0,unsigned int val1){
	unsigned int v0=val0;
	unsigned int v1=val1;
	unsigned int s0=0;

	for(unsigned int n=0; n<N; n++){
		s0+=0x9e3779b9;
		v0+=((v1<<4)+0xa341316c)^(v1+s0)^((v1>>5)+0xc8013ea4);
		v1+=((v0<<4)+0xad90777d)^(v0+s0)^((v0>>5)+0x7e95761e);
	}

	return v0;
}

// Generate random unsigned int in [0, 2^24)
static __device__ __inline__ unsigned int lcg(unsigned int &prev){
	const unsigned int LCG_A=1664525u;
	const unsigned int LCG_C=1013904223u;
	prev=(LCG_A * prev+LCG_C);
	return prev&0x00FFFFFF;
}

/*
static __host__ __device__ __inline__ unsigned int lcg2(unsigned int &prev){
	prev=(prev*8121+28411)%134456;
	return prev;
}
// Generate random float in [0, 1)
static __host__ __device__ __inline__ float rnd(unsigned int &prev){
	return ((float)lcg(prev)/(float)0x01000000);
}

static __host__ __device__ __inline__ unsigned int rot_seed(unsigned int seed,unsigned int frame){
	return seed ^ frame;
}
*/

/**
* 一维StratifiedSample抽样
* 分层抽样必须提前知道样本点有多少，不然没有办法划分区间
* "dec_dataContext.h"里有生成的样本总数参数V_C_DATA_CONTEXT
* "dec_frameNumber.h"里有样本的下标参数V_C_FRAME_NUMBER_UINT
*
* @param[in] index  表示样本下标
RT_FUNC_S void GetSample2_stratified(uint& seed,float2& out){
	const float interval=1.f/V_C_DATA_RENDERER.samples_per_pixel_col;
	float delta=rnd(seed);
	seed=rot_seed(seed,V_C_DATA_RENDERER.frame_current*2u);
	out.x=fminf(((int)V_C_DATA_RENDERER.frame_current/V_C_DATA_RENDERER.samples_per_pixel_col+delta)*interval,M_1_MINUS_EPSILON);

	delta=rnd(seed);
	seed=rot_seed(seed,V_C_DATA_RENDERER.frame_current*2u+1u);
	out.y=fminf(((int)V_C_DATA_RENDERER.frame_current % V_C_DATA_RENDERER.samples_per_pixel_row+delta)*interval,M_1_MINUS_EPSILON);
}
*/
}


RT_CALLABLE_PROGRAM uint CF_GETRANDOMSEED(uint index,uint frameCount){
	unsigned int s0=0;

	for(unsigned int n=0; n<16; n++){
		s0+=0x9e3779b9;
		index+=((frameCount<<4)+0xa341316c)^(frameCount+s0)^((frameCount>>5)+0xc8013ea4);
		frameCount+=((index<<4)+0xad90777d)^(index+s0)^((index>>5)+0x7e95761e);
	}

	return index;
}

/**
* Sample one float at range of [0,1)
*/
RT_CALLABLE_PROGRAM RandomFloat1 CF_GETSAMPLE1(uint& seed){
	return ((float)lcg(seed)/(float)0x01000000);
}

/*
* Sample two float at range of [0,1)
*/
RT_CALLABLE_PROGRAM RandomFloat2 CF_GETSAMPLE2(uint& seed){
	RandomFloat2 samples;
	samples.x=((float)lcg(seed)/(float)0x01000000);
	samples.y=((float)lcg(seed)/(float)0x01000000);
	return samples;
}

#pragma endregion

#pragma region DISTRIBUTION

/**
* endIndex 是不能返回的，就像正常的数组一样。
* 返回的值是在整体上的索引，并非在startIndex基础上；
*/
RT_FUNC_F int FindLowBoundOffset_(int BufferID,int startIndex,int endIndex,float value){
	if(value<=0.f)return startIndex;
	if(value>=1.f)return endIndex-1;

	int mid=startIndex+1;
	while(startIndex<endIndex-1){
		mid=(int)((startIndex+endIndex)*.5f);
		float v=rtBufferId<float,1>(BufferID)[mid];
		if(v>value)endIndex=mid;
		if(v<=value)startIndex=mid;
	}
	//rtPrintf("start[%d],end[%d],mid[%d],v[%f]",startIndex,endIndex,mid,rtBufferId<float,1>(BufferID)[mid]);
	return mid-1;
}

RT_CALLABLE_PROGRAM void CF_DISTRIBUTION1D_SAMPLEDISCRETE(const SData_Light& data,RandomFloat1 rf,int& offset,float& PDF){
	int CDFBufferID=data.meshAreaCDFBufferID;
	float integration=data.meshAreaIntegration;
	int valueBufferID=data.meshAreaBufferID;
	int size=data.meshCount;
	//Assert(rf<1.0f);
	offset=FindLowBoundOffset_(CDFBufferID,0,size+1,rf);
	//Assert(offset<=size);

	float v=rtBufferId<float,1>(valueBufferID)[offset];
	PDF=v/(integration*size);
}


/*
RT_FUNC void SampleDiscrete(const RandomFloatSample2& rs,UV& uv,float& PDF){
//Assert(rs.x<1.0f && rs.y<1.0f);
int off_y=FindLowBoundOffset_(MarginalCDFBufferID,0,size.y+1,rs.y);
uv.y=float(off_y)/float(size.y);
//Assert(uv.y<=1.f);

int startIndex=off_y*(size.x+1);
int endIndex=(off_y+1)*(size.x+1);
int off_x=FindLowBoundOffset_(CDFBufferID,startIndex,endIndex,rs.x);
uv.x=float(off_x-startIndex)/float(size.x);
//Assert(uv.x<=1.f);

PDF=GETLVALUE(off_x)/GETIVALUE(size.y);

}
*/
RT_CALLABLE_PROGRAM void CF_DISTRIBUTION2D_SAMPLECONTINUOUS(const SData_Light& data,const RandomFloat2& rs,RandomVariable2& uv,float& PDF){
	uint2 size=data.HDR_size;
	int CDFBufferID=data.HDR_CDFBufferID;
	int MarginalCDFBufferID=data.HDR_MarginalCDFBufferID;
	int IntegrationBufferID=data.HDR_IntegrationBufferID;
	int LuminanceBufferID=data.HDR_LuminanceBufferID;

	//Assert(rs.x < 1.0f && rs.y < 1.0f);
	int off_y=FindLowBoundOffset_(MarginalCDFBufferID,0,size.y+1,rs.y);
	//rtPrintf("wow!CDFBufferID[%d],MarginalCDFBufferID[%d],IntegrationBufferID[%d],LuminanceBufferID[%d]",CDFBufferID,MarginalCDFBufferID,IntegrationBufferID,LuminanceBufferID);

	//discrete sample only:
	//float pdf_y=GETIVALUE(off_y)/GETIVALUE(size.y);

	//continues sampling;
#define GETMCDFVALUE(index)(rtBufferId<float,1>(MarginalCDFBufferID)[index])
	float du=(rs.y-GETMCDFVALUE(off_y))/(GETMCDFVALUE(off_y+1)-GETMCDFVALUE(off_y));
#undef GETMCDFVALUE

	// 下面没有使用公式uv.y=1.f-(off_y+du)/(size.y+1.f)计算uv.y，
	// 是因为图像本身的Y轴是颠倒的。
	uv.y=(off_y+du)/float(size.y);

	const int startIndex=off_y*(size.x+1);
	const int endIndex=(off_y+1)*(size.x+1);
	int off_x=FindLowBoundOffset_(CDFBufferID,startIndex,endIndex,rs.x);
#if 0
	rtPrintf("off_y off_x is:%d,%d\n",off_y,off_x-startIndex);
#endif

	//discrete sample only:float pdf_x=GETLVALUE(off_x)/GETIVALUE(off_y);
#define GETCDFVALUE(index)(rtBufferId<float,1>(CDFBufferID)[index])
	float dv=(rs.x-GETCDFVALUE(off_x))/(GETCDFVALUE(off_x+1)-GETCDFVALUE(off_x));
	uv.x=(off_x-startIndex+dv)/(size.x);
#undef GETCDFVALUE

	//PDF=pdf_y*pdf_x;
	PDF=rtBufferId<float,1>(LuminanceBufferID)[off_x]/rtBufferId<float,1>(IntegrationBufferID)[size.y];
};

RT_CALLABLE_PROGRAM float CF_DISTRIBUTION2D_PDF(const SData_Light& data,UV2& uv){
	uint2 size=data.HDR_size;
	int IntegrationBufferID=data.HDR_IntegrationBufferID;
	int LuminanceBufferID=data.HDR_LuminanceBufferID;
	//Assert(0.f <= uv.x < 1.0f);
	//Assert(0.f <= uv.y < 1.0f);
	int iu=uv.x*size.x;
	int iv=uv.y*size.y;
	int index=iv*size.x+iu;
	return rtBufferId<float,1>(LuminanceBufferID)[index]/rtBufferId<float,1>(IntegrationBufferID)[size.y];

}

struct PhongDistribution{
public:

	RT_FUNC float D(const LMD& data,const Nor& wh_g) const{
		const float costhetah=fabsf(dot(wh_g,data.sn_g));

		const float d=1.f-costhetah * costhetah;
		if(d<=0.f) return 0.f;
		const float HdotT=dot(wh_g,data.tangent_g);
		const float HdotB=dot(wh_g,data.bitangent_g);
		const float e=(data.anisotropicUV.x * HdotT*HdotT+data.anisotropicUV.y * HdotB*HdotB)/d;
		const float r=sqrtf((data.anisotropicUV.x+2.f) * (data.anisotropicUV.y+2.f)) * M_1_2PIf * powf(costhetah,e);
		return r;
	}

	RT_FUNC float Pdf(const LMD& data,const Nor& wh_g) const{
		// Compute PDF for $\wi$ from anisotropic distribution
		const float costhetah=fabsf(dot(wh_g,data.sn_g));
		const float ds=1.f-costhetah * costhetah;
		if(ds>0.f && dot(data.wo_g,wh_g)>0.f){
			const float HdotT=dot(wh_g,data.tangent_g);
			const float HdotB=dot(wh_g,data.bitangent_g);
			const float e=(data.anisotropicUV.x * HdotT*HdotT+data.anisotropicUV.y *HdotB*HdotB)/ds;
			const float d=sqrtf((data.anisotropicUV.x+1.f) * (data.anisotropicUV.y+1.f)) * M_1_2PIf * powf(costhetah,e);
			return d;
		}
		return 0.f;
	}


	RT_FUNC_F void Sample_wh(const LMD&data,RandomVariable2& u,Nor& wh_g) const{
		// Sample from first quadrant and remap to hemisphere to sample $\wh$
		float phi,costheta;
		if(u.x<.25f){
			u.x*=4.f;
			sampleFirstQuadrant(data,u,phi,costheta);
		} else if(u.x<.5f){
			u.x=4.f * (.5f-u.x);
			sampleFirstQuadrant(data,u,phi,costheta);
			phi=M_PIf-phi;
		} else if(u.x<.75f){
			u.x=4.f * (u.x-.5f);
			sampleFirstQuadrant(data,u,phi,costheta);
			phi+=M_PIf;
		} else{
			u.x=4.f * (1.f-u.x);
			sampleFirstQuadrant(data,u,phi,costheta);
			phi=M_2PIf-phi;
		}
		float sintheta=sqrtf(fmaxf(0.f,1.f-costheta*costheta));
		wh_g=SphericalDirection(sintheta,costheta,phi);
		wh_g=normalize(wh_g);

		float3 result=wh_g.x*data.tangent_g+wh_g.y*data.bitangent_g+wh_g.z*data.sn_g;
		if(!SameHemisphere(result,data.wo_g,data.sn_g)){ result*=-1.f; }
		wh_g=result;
	}

	RT_FUNC void sampleFirstQuadrant(const LMD& data,const RandomVariable2& u,float& phi,float& costheta) const{
		if(data.anisotropicUV.x==data.anisotropicUV.y) phi=M_PI_2f * u.x;
		else phi=atanf(sqrtf((data.anisotropicUV.x+1.f)/(data.anisotropicUV.y+1.f)) * tanf(M_PI_2f * u.x));
		const float cosphi=cosf(phi);
		const float sinphi=sinf(phi);
		costheta=powf(u.y,1.f/(data.anisotropicUV.x * cosphi * cosphi+data.anisotropicUV.y * sinphi * sinphi+1.f));
	}

};

#pragma endregion

#pragma region TEXTURE
RT_CALLABLE_PROGRAM Color4 CF_GETTEXTURE_GLOSSY(int bufId,int offset,UV2 uv){
	SData_Texture& data=rtBufferId<SData_Material,1>(bufId)[offset].BS_textureGlossy;
	if(data.enable){
		uv.x*=data.UVrepeat.x;
		uv.y*=data.UVrepeat.y;
		if(fabsf(data.rotation)>M_MIN_FLOAT){
			const float c=cos(data.rotation);
			const float s=sin(data.rotation);
			const float temp=c*uv.x-s*uv.y;
			uv.y=s*uv.x+c*uv.y;
			uv.x=temp;
		}
		Color4 c=rtTex2D<float4>(data.TSID,uv.x,uv.y);
		c.x=powf(c.x,data.invGamma);
		c.y=powf(c.y,data.invGamma);
		c.z=powf(c.z,data.invGamma);
		c.w=1.f;
		return c;
	} else return data.color;
}

RT_CALLABLE_PROGRAM Color4 CF_GETTEXTURE_DIFFUSE(int bufId,int offset, UV2 uv){
	SData_Texture& data=rtBufferId<SData_Material,1>(bufId)[offset].BS_textureDiffuse;
	if(data.enable){
		uv.x*=data.UVrepeat.x;
		uv.y*=data.UVrepeat.y;
		if(fabsf(data.rotation)>M_MIN_FLOAT){
			const float c=cos(data.rotation);
			const float s=sin(data.rotation);
			const float temp=c*uv.x-s*uv.y;
			uv.y=s*uv.x+c*uv.y;
			uv.x=temp;
		}
		Color4 c=rtTex2D<float4>(data.TSID,uv.x,uv.y);
		c.w=1.f;
		return c;
	} else return data.color;
}

RT_CALLABLE_PROGRAM Nor CF_GETTEXTURE_NORMAL(const SData_Texture& data, Intersection aInset){
	if(data.enable){
		aInset.texCoord.x*=data.UVrepeat.x;
		aInset.texCoord.y*=data.UVrepeat.y;
		if(fabsf(data.rotation)>M_MIN_FLOAT){
			const float c=cos(data.rotation);
			const float s=sin(data.rotation);
			const float temp=c*aInset.texCoord.x-s*aInset.texCoord.y;
			aInset.texCoord.y=s*aInset.texCoord.x+c*aInset.texCoord.y;
			aInset.texCoord.x=temp;
		}
		Color4 c=rtTex2D<float4>(data.TSID,aInset.texCoord.x,aInset.texCoord.y);
		c.w=1.f;
		Nor nor=make_float3(F_COLOR_TO_NORMAL(c));
		nor=nor.x*aInset.tbn.Tangent+nor.y*aInset.tbn.Bitangent+nor.z*aInset.tbn.Normal;
		nor=lerp(nor,aInset.geoNor,data.bumpscale+1.f);
		return nor;
	} else return aInset.geoNor;
}

RT_CALLABLE_PROGRAM Radiance4 CF_GETSKYBOXRADIANCE(int TSID,const Dir& dir_g){
	if(TSID<0)return make_float4(0.f);

	UV2 uv;
	float phi=atan2f(dir_g.x,dir_g.z);
	uv.x=phi*M_1_2PIf;
	float theta=acos(dir_g.y);
	uv.y=1.f-theta*M_1_PIf;

	return rtTex2D<float4>(TSID,uv.x,uv.y);
}

/**
* 调用该函数需要注意的是：球谐函数使用的是标准的右手纹理坐标系，需要将wi_g转化
* 另一个需要注意的是：通过freeimage加载的图片是上下颠倒的；
* 内容的实现依据以下论文：
* https://cseweb.ucsd.edu/~ravir/papers/envmap/envmap.pdf
*/
RT_CALLABLE_PROGRAM Radiance4 CF_GETRADIANCEFROMSH(int IdOfSHCbuffer,Nor wi_g){

	float tmpy=wi_g.y,tmpx=wi_g.x;
	wi_g.x=wi_g.z;
	wi_g.y=tmpx;
	wi_g.z=-tmpy;


#define GETSHCOF(idx) rtBufferId<float3, 1>(IdOfSHCbuffer)[idx]

#if 0
	float3 value=GETSHCOF(4);
	rtPrintf("wow,wi_g[%f,%f,%f],ldtype[%d],value[%f,%f,%f],,",wi_g.x,wi_g.y,wi_g.z,ld.lightType,value.x,value.y,value.z);
#endif

	Radiance3 Li=make_float3(0.f);
	Li+=GETSHCOF(0u)*Y00;
	Li+=GETSHCOF(1u)*Y1_1(wi_g.y);
	Li+=GETSHCOF(2u)*Y10(wi_g.z);
	Li+=GETSHCOF(3u)*Y11(wi_g.x);
	Li+=GETSHCOF(4u)*Y2_2(wi_g.x,wi_g.y);
	Li+=GETSHCOF(5u)*Y2_1(wi_g.y,wi_g.z);
	Li+=GETSHCOF(6u)*Y20(wi_g.z);
	Li+=GETSHCOF(7u)*Y21(wi_g.x,wi_g.z);
	Li+=GETSHCOF(8u)*Y22(wi_g.x,wi_g.y);

#undef GETSHCOF

	return make_float4(Li,1.f);
}

RT_CALLABLE_PROGRAM float CF_GETENVRADIANCE(int TSID,int bufferID,const Nor& rayDir,float blend,Color4& outColor){
	int N=rtBufferId<SData_Light,1>(bufferID).size();
	for(uint i=0u; i<N; ++i){
		const SData_Light& ld=rtBufferId<SData_Light,1>(bufferID)[i];
		if(ld.lightFuncIndex==CFINDEX_LIGHT_ENVIRONMENT){
			Radiance4 l=CF_GETSKYBOXRADIANCE(TSID,rayDir);
			Radiance4 sh=CF_GETRADIANCEFROMSH(ld.SH_cBufferID,rayDir)*ld.energyBalance;
			Radiance4 L=lerp(sh,l,blend);
			const float& F{ld.HDR_inv_gamma};
			outColor=make_float4(powf(L.x,F),powf(L.y,F),powf(L.z,F),1.0f);;
			return ld.HDR_exposure;
		}
	}
	return 0.f;
}

RT_CALLABLE_PROGRAM Nor CF_GETTRIPLANENORMAL(int TSID,const Pos& pos,const Nor& gn){
	/**
	* 参考资料
	https://medium.com/@bgolus/normal-mapping-for-a-triplanar-shader-10bf39dca05a
	*/

	Nor norX=make_float3(F_COLOR_TO_NORMAL(rtTex2DLod<float4>(TSID,pos.z,pos.y,0)));
	Nor norY=make_float3(F_COLOR_TO_NORMAL(rtTex2DLod<float4>(TSID,pos.x,pos.z,0)));
	Nor norZ=make_float3(F_COLOR_TO_NORMAL(rtTex2DLod<float4>(TSID,pos.x,pos.y,0)));

	if(gn.x<0.f) norX.z*=-1.f;
	if(gn.y<0.f) norY.z*=-1.f;
	if(gn.z<0.f) norZ.z*=-1.f;

#define F 2.f
	float3 blend=make_float3(powf(gn.x,F),powf(gn.y,F),powf(gn.z,F));
	blend/=dot(blend,make_float3(1.f));

	float3 X=make_float3(norX.z,norX.y,norX.x)*blend.x;
	float3 Y=make_float3(norY.x,norY.z,norY.y)*blend.y;
	float3 Z=make_float3(norZ.x,norZ.y,norZ.z)*blend.z;
	return normalize(X+Y+Z);
#undef F


};
#pragma endregion

#pragma region LIGHTMODEL

RT_FUNC_F float FrCond(float cosi){
	//折射率
	const float eta=0.2f;
	//吸收系数
	const float k=3.f;
	const float tmp=(eta*eta+k*k) * cosi*cosi;
	const float Rparl2=(tmp-(2.f * eta * cosi)+1)/(tmp+(2.f * eta * cosi)+1);
	const float tmp_f=eta*eta+k*k;
	const float Rperp2=(tmp_f-(2.f * eta * cosi)+cosi*cosi)/(tmp_f+(2.f * eta * cosi)+cosi*cosi);
	return (Rparl2+Rperp2)*.5f;
}

RT_FUNC_F float D(const Nor &wh,const Nor& n_g,float roughness){
	const float costhetah=fabsf(dot(n_g,wh));
	return roughness * M_1_2PIf* powf(costhetah,roughness);
}

RT_FUNC_F  float G(const Nor &wo,const Nor& n_g,const Nor &wi,const Nor &wh){
	const float tmp=2.f*fabsf(dot(n_g,wh)/dot(wo,wh));
	const float NdotWo=fabsf(dot(n_g,wo));
	const float NdotWi=fabsf(dot(n_g,wi));
	return min(1.f,min(NdotWo*tmp,NdotWi*tmp));
}

#pragma region Lambertian
RT_CALLABLE_PROGRAM void Lambertian_Pdf(LMD& data,uint& seed){
	//RT_CALLABLE_PROGRAM float Lambertian_Pdf(const Nor& n_g,const Nor& wo_g,const Nor& wi_g){
		/*
		 * 这里需要注意，由于计算机精度问题导致随机的半球法线不可能完美的被限制在半球内部，所以导致SampleHemisphiere返回false
		 * 而整个Lambertian_Pdf函数返回0.f
		 * 而0.f的话由于存在除以0.f的逻辑，从而导致结果无穷大。需要注意；
		 * 所以下面的逻辑不能用
		return SameHemisphere(wo_g,wi_g,prd.n_g)?fabsf(dot(wi_g,prd.n_g)) * M_1_PIf:0.f;
		*/
	data.PDF=fabsf(dot(data.wi_g,data.sn_g)) * M_1_PIf;
}

RT_CALLABLE_PROGRAM void Lambertian_f(LMD& data,uint& seed){ data.BRDF=M_1_PIf; }
//RT_CALLABLE_PROGRAM float Lambertian_f(const Nor& n_g,const Nor& wo_g,const Nor& wi_g,float roughness){ return M_1_PIf; }

RT_CALLABLE_PROGRAM void Lambertian_Sample_f(LMD& data,uint& seed){
	//RT_CALLABLE_PROGRAM float Lambertian_Sample_f(const Nor& n_g,const Nor& wo_g,Nor& wi_g,float& pdf,uint& seed,float roughness){
	if(dot(data.sn_g,data.wo_g)<0.f)data.wi_g=data.sn_g;
	else{
		Onb onb(data.sn_g);
		RandomVariable2 samples=CF_GETSAMPLE2(seed);
		data.wi_g=CF_COSINESAMPLEHEMISPHERE_QUICKER(samples,0.f,1.f);
		/**
		* 目前不做图元双面渲染，原因是
		* 1 真实世界中不存在数学中的2D平面图元；
		* 2 消耗性能；
		* 3 无法良好判断图元的正反面；
		if(dot(wo_g,n_g)<0.f)wi_g.z*=-1.f;
		*/
		onb.inverse_transform(data.wi_g);
		data.NormalizeWi();
	}
	Lambertian_Pdf(data,seed);
	Lambertian_f(data,seed);
}

RT_CALLABLE_PROGRAM void Lambertian_Albedo(LMD& data){
	//RT_CALLABLE_PROGRAM float Lambertian_Albedo(const Nor& nor,const Nor& wo_g,const Nor& wi_g){
	data.albedo=M_1_PIf;
}
#pragma endregion

#pragma region TorranceSparrow
RT_CALLABLE_PROGRAM void TorranceSparrow_Pdf(LMD& data,uint& seed){
	//RT_CALLABLE_PROGRAM float TorranceSparrow_Pdf(const Nor& n_g,const Nor& wo_g,const Nor& wi_g){
	// TODO:这个PDF计算是错的；
	data.PDF=data.WoWiAtSamehemisphere()?fabsf(dot(data.wi_g,data.sn_g))*M_1_PIf:0.f;
}

RT_CALLABLE_PROGRAM void TorranceSparrow_f(LMD& data,uint& seed){
	//RT_CALLABLE_PROGRAM float TorranceSparrow_f(const Nor& n_g,const Nor& wo_g,const Nor& wi_g,float roughness){
	const float cosTheta=fabsf(data.WiDotN()*data.WoDotN());
	if(cosTheta<=M_EPSILON) data.BRDF=0.f;
	else{
		Nor wh_g=data.GetWh_g();
		if(dot(wh_g,wh_g)<=M_EPSILON) data.BRDF=0.f;
		else{
			data.BRDF=D(wh_g,data.sn_g,data.roughness)*G(data.wo_g,data.sn_g,data.wi_g,wh_g)*FrCond(dot(data.wi_g,wh_g))/(4.f * cosTheta);
		}
	}
}

RT_CALLABLE_PROGRAM void TorranceSparrow_Sample_f(LMD& data,uint& seed){
	//RT_CALLABLE_PROGRAM float TorranceSparrow_Sample_f(const Nor& n_g,const Nor& wo_g,Nor& wi_g,float& pdf,uint& seed,float roughness){
		//roughness [1~ +inf)
	RandomVariable2 samples=CF_GETSAMPLE2(seed);
	if(samples.x<0.01f)samples.x+=0.01f;
	const float costheta=powf(samples.x,1.f/data.roughness);
	const float sintheta=sqrtf(max(0.f,1.f-costheta*costheta));
	const float phi=samples.y * M_2PIf;
	Nor wh_g=SphericalDirection(sintheta,costheta,phi);
	Onb onb(data.sn_g);
	onb.inverse_transform(wh_g);

	data.wi_g=normalize(reflect(-data.wo_g,wh_g));
	// Compute PDF for $\wi$ from Blinn distribution
	if(!data.WoWiAtSamehemisphere()){
		data.wi_g*=-1.f;
	}
	////////////////////////////////////////////////////////////////////////////////////////////////////
	//pdf=(roughness * powf(costheta,roughness))/(M_8PIf *  fabsf(dot(wo_g,wh_g)));
	// 优化上面为下一行：
	data.PDF=(data.roughness * samples.x)/(M_8PIf *  fabsf(dot(data.wo_g,wh_g)));
	////////////////////////////////////////////////////////////////////////////////////////////////////
	TorranceSparrow_f(data,seed);
}

RT_CALLABLE_PROGRAM void TorranceSparrow_Albedo(LMD& data){
	//RT_CALLABLE_PROGRAM float TorranceSparrow_Albedo(const Nor& n_g,const Nor& wo_g,const Nor& wi_g){
	data.albedo=1.f;
}
#pragma endregion

#pragma region SpecularReflect
RT_CALLABLE_PROGRAM void SpecularReflect_Pdf(LMD& data,uint& seed){
	data.PDF=0.f;
}

RT_CALLABLE_PROGRAM void SpecularReflect_f(LMD& data,uint& seed){
	data.BRDF=0.f;
}

RT_CALLABLE_PROGRAM void SpecularReflect_Sample_f(LMD& data,uint& seed){
	data.wi_g=reflect(-data.wo_g,data.sn_g);
	data.NormalizeWi();
	data.PDF=1.f;
	data.BRDF=1.f/fabsf(data.WiDotN());
}

RT_CALLABLE_PROGRAM void SpecularReflect_Albedo(LMD& data,uint& seed){
	data.albedo=1.f;
}
#pragma endregion

#pragma region SpecularRefract
RT_CALLABLE_PROGRAM void SpecularRefract_Pdf(LMD& data,uint& seed){
	data.PDF=0.f;
}

RT_CALLABLE_PROGRAM void SpecularRefract_f(LMD& data,uint& seed){
	data.BRDF=0.f;
}

RT_CALLABLE_PROGRAM void SpecularRefract_Sample_f(LMD& data,uint& seed){
	if(!refract(data.wi_g,-data.wo_g,data.sn_g,data.IOR)){
		//全反射
		data.wi_g=reflect(-data.wo_g,data.sn_g);
	}
	data.NormalizeWi();
	data.PDF=1.f;
	data.BRDF=1.f/fabsf(data.WiDotN());
}

RT_CALLABLE_PROGRAM void SpecularRefract_Albedo(LMD& data){
	data.albedo=1.f;
}
#pragma endregion

#pragma region Ashikhmin-Shirley
RT_CALLABLE_PROGRAM void AshikminShirley_Pdf(LMD& data,uint& seed){
	const Nor wh_g=data.GetWh_g();
	PhongDistribution dtrb;

	// 下面是平衡自适应，乘以0.5f的意思是给分子乘以2（pdf做分母）具体参考平衡自适应，如果
	// 采用幂自适应，收敛会更快；
	data.PDF=.5f * (fabsf(data.WiDotN())* M_1_PIf+dtrb.Pdf(data,wh_g)/(4.f*dot(data.wo_g,wh_g)));
}

RT_CALLABLE_PROGRAM void AshikminShirley_f(LMD& data,uint& seed){
	//diffuse partial
	const float diffuse=(28.f/(23.f*M_PIf)) * data.reflatance.x * (1.f-data.reflatance.y) *(1.f-powf(1.f-.5f * fabsf(data.WiDotN()),5.f)) *(1.f-powf(1.f-.5f * fabsf(data.WoDotN()),5.f));
	Nor wh_g=data.GetWh_g();
	if(length(wh_g)==0) data.BRDF=0.f;
	else{

		//specular partial
		const float maxFactor=fmaxf(fmaxf(fabsf(data.WiDotN()),fabsf(data.WoDotN())),M_MIN_RADIANCE);
		const float absTheta=fmaxf(fabsf(dot(wh_g,data.wi_g)),M_MIN_RADIANCE);
		const float f=fresnel_schlick(absTheta,5.f,data.reflatance.y,1.f);

		PhongDistribution dtrb;
		const float specular=dtrb.D(data,wh_g)*f/(4.f * absTheta * maxFactor);

		if(data.WoWiAtSamehemisphere()){
			data.BRDF=data.reflatance.x*diffuse+data.reflatance.y*specular;
		} else{
			data.BRDF=0.f;
		}
	}
}

RT_CALLABLE_PROGRAM void AshikminShirley_Sample_f(LMD& data,uint& seed){
	RandomFloat2 u=GetSample2(seed);
	if(u.x<0.5f){
		u.x*=2.f;
		// Cosine-sample the hemisphere, flipping the direction if necessary
		data.wi_g=CosineSampleHemisphere(u.x,u.y);
		if(data.WoDotN()<0.f)data.wi_g.z*=-1.f;
		Onb onb(data.sn_g);
		onb.inverse_transform(data.wi_g);
		data.NormalizeWi();
	} else{
		u.x=2.f * (u.x-.5f);
		Nor wh_g;
		PhongDistribution dtrb;
		dtrb.Sample_wh(data,u,wh_g);
		data.wi_g=reflect(-data.wo_g,wh_g);

		if(!data.WoWiAtSamehemisphere()){
			data.BRDF=0.f;
			return;
		}
	}
	AshikminShirley_Pdf(data,seed);
	AshikminShirley_f(data,seed);
}

RT_CALLABLE_PROGRAM void AshikminShirley_Albedo(LMD& data){
	data.albedo=1.f;
}
#pragma endregion

#pragma endregion

#pragma region LIGHTS
RT_FUNC_S float Falloff(float costheta,float cosTotalWidth,float cosFalloffStart){
	if(costheta<cosTotalWidth||costheta<=0.f)return 0.;
	if(costheta>cosFalloffStart)return 1.;
	// Compute falloff inside spotlight cone
	float delta=(costheta-cosTotalWidth)/(cosFalloffStart-cosTotalWidth);
	return delta*delta*delta*delta;
}
RT_FUNC_S void CalcWithMatrix(float3& p,int id){
	float3 tmpp=p;
	p.x=tmpp.x*GET_F_1D(id,0)+tmpp.y*GET_F_1D(id,1)+tmpp.z*GET_F_1D(id,2)+GET_F_1D(id,3);
	p.y=tmpp.x*GET_F_1D(id,4)+tmpp.y*GET_F_1D(id,5)+tmpp.z*GET_F_1D(id,6)+GET_F_1D(id,7);
	p.z=tmpp.x*GET_F_1D(id,8)+tmpp.y*GET_F_1D(id,9)+tmpp.z*GET_F_1D(id,10)+GET_F_1D(id,11);
}
RT_FUNC_S void UVToNor(const UV2& uv,Nor& v,float& sintheta){
	const float theta=(1.f-uv.y)*M_PIf;
	const float phi=uv.x*M_2PIf;

	v.y=cosf(theta);
	sintheta=sinf(theta);
	v.x=sintheta*sinf(phi);
	v.z=sintheta*cosf(phi);
}

/**
* 使用的前提是HDR贴图没有旋转。
*/
RT_FUNC_S void DirToUV(const Dir& dir,UV2& uv){
	float phi=atan2f(dir.x,dir.z);
	uv.x=phi*M_1_2PIf;
	float theta=acos(dir.y);
	uv.y=1-theta*M_1_PIf;
}

#pragma region POINT_LIGHTS
RT_CALLABLE_PROGRAM void PointLight_Sample_L(const SData_Light& data,LMD& lmd,uint& seed){
	//RT_CALLABLE_PROGRAM Radiance4 PointLight_Sample_L(const SData_Light& data,const Pos& pos_g,Dir& wi_g,float& pdf,uint& seed){
	lmd.wi_g=normalize(data.pos-lmd.hitPos_g);
	lmd.PDF=1.f;
	lmd.L=data.diffuse*data.intensity/DistanceSquared(data.pos,lmd.hitPos_g);
}

RT_CALLABLE_PROGRAM Radiance4 PointLight_Power(const SData_Light& data){ return M_4PIf*data.diffuse*data.intensity; }

RT_CALLABLE_PROGRAM float PointLight_Pdf(const SData_Light&,const Pos &p,const Dir &wi){ return 0.f; }
//RT_FUNC float3 Le(const RayDifferential &r) const;
//RT_FUNC void SHProject(const Pos &p,float pEpsilon,int lmax,float3 *coeffs){ }
#pragma endregion

#pragma region SPOT_LIGHTS
RT_CALLABLE_PROGRAM void SpotLight_Sample_L(const SData_Light& data,LMD& lmd,uint& seed){
	//RT_CALLABLE_PROGRAM float4 SpotLight_Sample_L(const SData_Light& data,const Pos& pos_g,Dir& wi_g,float& pdf,uint& seed){
	lmd.wi_g=normalize(data.pos-lmd.hitPos_g);
	lmd.PDF=1.f;
	float cosTotalWidth=cosf(F_RADIAN(data.totalTheta));
	float cosFalloffStart=cosf(F_RADIAN(data.fullLightTheta));
	float cosTheta=dot(-lmd.wi_g,data.dir);
	lmd.L=data.diffuse*data.intensity * Falloff(cosTheta,cosTotalWidth,cosFalloffStart)/DistanceSquared(data.pos,lmd.hitPos_g);
}
RT_CALLABLE_PROGRAM Radiance4 SpotLight_Power(const SData_Light& data){ return M_4PIf*data.diffuse*data.intensity; }

/**
* 针对未与场景几何体对象碰撞的光线，无穷远区域光源须向其“贡献”辐射度。对此，可向Light基类加入一方法，沿未碰撞光线并根据当前光源返回发射辐射度。
* 针对其他类型光源，默认实现并不返回辐射度。
*/
RT_CALLABLE_PROGRAM float SpotLight_Pdf(const SData_Light&,const Pos &p,const Dir &wi){ return 0.f; }
//RT_FUNC float3 Le(const RayDifferential &r) const;
//RT_FUNC void SHProject(const Pos &p,float pEpsilon,int lmax,float3 *coeffs){ }
#pragma endregion

#pragma region DISTANT_LIGHTS
RT_CALLABLE_PROGRAM void DistantLight_Sample_L(const SData_Light& data,LMD& lmd,uint& seed){
	//RT_CALLABLE_PROGRAM Irradiance4 DistantLight_Sample_L(const SData_Light& data,const Pos& pos_g,Dir& wi_g,float& pdf,uint& seed){
	lmd.wi_g=-data.dir;
	lmd.PDF=1.f;
	lmd.L=data.diffuse*data.intensity;
}

RT_CALLABLE_PROGRAM float4 DistantLight_Power(){ return make_float4(0.f); }

/**
* 针对未与场景几何体对象碰撞的光线，无穷远区域光源须向其“贡献”辐射度。对此，可向Light基类加入一方法，沿未碰撞光线并根据当前光源返回发射辐射度。
* 针对其他类型光源，默认实现并不返回辐射度。
*/
RT_CALLABLE_PROGRAM float DistantLight_Pdf(const SData_Light&,const Pos &p,const Dir &wi){ return 0.f; }
//RT_FUNC float3 Le(const RayDifferential &r) const;
//RT_FUNC void SHProject(const Pos &p,float pEpsilon,int lmax,float3 *coeffs){ }
#pragma endregion

#pragma region AREA_LIGHTS
/**
* 调用者负责传递场景内一点的世界空间位置，以及光照采样的时刻，最终返回到达该点的辐射度。
* 该函数还负责初始化相对于光源的入射方向wi，以及初始化VisibilityTester对象；
* 如果光源不是点光源，该方法需要对光源表面上的一点进行随机采样。
*/
RT_CALLABLE_PROGRAM void AreaLight_Sample_L(const SData_Light& data,LMD& lmd,uint& seed){
	//RT_CALLABLE_PROGRAM Irradiance4 AreaLight_Sample_L(const SData_Light& data,const Pos& pos_g,Dir& wi_g,float& PDF,uint& seed){

	int offset;
	float mapPDF;
	RandomFloat1 component=CF_GETSAMPLE1(seed);
	CF_DISTRIBUTION1D_SAMPLEDISCRETE(data,component,offset,mapPDF);

	// local space;
	const int3 v_idx=GET_I3_1D(data.bufferID_index,offset);
	Pos p0=GET_F3_1D(data.bufferID_vertex,v_idx.x);
	Pos p1=GET_F3_1D(data.bufferID_vertex,v_idx.y);
	Pos p2=GET_F3_1D(data.bufferID_vertex,v_idx.z);

	//to world space;
	//**************************************************start**************************************************
	// 下面函数导致将p0,p1,p2点转化为了当前mesh的坐标，从而出现错误。
	//p0=rtTransformPoint(RT_OBJECT_TO_WORLD,p0);
	//p1=rtTransformPoint(RT_OBJECT_TO_WORLD,p1);
	//p2=rtTransformPoint(RT_OBJECT_TO_WORLD,p2);
	// 应该为：
	CalcWithMatrix(p0,data.bufferID_worldMatrix);
	CalcWithMatrix(p1,data.bufferID_worldMatrix);
	CalcWithMatrix(p2,data.bufferID_worldMatrix);
	//***************************************************end*************************************************
	/*
	rtPrintf("%f,%f,%f\n",p0.x,p0.y,p0.z);
	rtPrintf("%f,%f,%f\n",p1.x,p1.y,p1.z);
	rtPrintf("%f,%f,%f\n",p2.x,p2.y,p2.z);
	*/

	RandomFloat2 ls=CF_GETSAMPLE2(seed);
	RandomVariable1 u,v;
	CF_UNIFORMSAMPLETRIANGLE(ls.x,ls.y,u,v);
	// centric coord.
	Pos p=u*p0+v*p1+(1.f-u-v)*p2;
	lmd.wi_g=p-lmd.hitPos_g;
	Nor n=normalize(cross(p1-p0,p2-p0));

	//**************************************************start**************************************************
	// 下面不做点积判断，因为那样会对可见区域过小的区域光造成大量采样丢失；从而噪点无法收敛；
	//if(dot(n,-wi_g)>0){
	//same side;
	//***************************************************end*************************************************
	const float dis=length(lmd.wi_g);
	//**************************************************start**************************************************
	// 以前这里有单位化，去掉的原因是外部需要知道wi_g的长度，从而发射阴影查询射线；
	//wi_g/=dis;
	//***************************************************end*************************************************

	// calc PDF;
	const float area=rtBufferId<float,1>(data.meshAreaBufferID)[offset];

	//**************************************************start**************************************************
	// 下面这句
	//PDF=(dis*dis)/((AbsCosTheta(-wi_g,n)*area)/dis);
	//优化为：
	lmd.PDF=(dis*dis*dis)/(fabsf(dot(lmd.wi_g,n))*area);
	//***************************************************end*************************************************

	const SData_Material& md=rtBufferId<SData_Material,1>(data.bufferID_materials)[data.bufferOffset_materials];
	lmd.L=md.BS_textureDiffuse.color*md.BS_AOIntensity/(dis*dis);
}

RT_CALLABLE_PROGRAM float4 AreaLight_Power(){ return make_float4(0.f);/*TODO:return L*M_PIf*data.sumArea;*/ }

/**
* 针对未与场景几何体对象碰撞的光线，无穷远区域光源须向其“贡献”辐射度。对此，可向Light基类加入一方法，沿未碰撞光线并根据当前光源返回发射辐射度。
* 针对其他类型光源，默认实现并不返回辐射度。
*/
RT_CALLABLE_PROGRAM float AreaLight_Pdf(const SData_Light&,const Pos &p,const Dir &wi){ return 0.f; }
//RT_FUNC float3 Le(const RayDifferential &r) const;
//RT_FUNC void SHProject(const Pos &p,float pEpsilon,int lmax,float3 *coeffs){ }
#pragma endregion

#pragma region ENV_LIGHTS
/**
* 调用者负责传递场景内一点的世界空间位置，以及光照采样的时刻，最终返回到达该点的辐射度。
* 该函数还负责初始化相对于光源的入射方向wi，以及初始化VisibilityTester对象；
* 如果光源不是点光源，该方法需要对光源表面上的一点进行随机采样。
*/
RT_CALLABLE_PROGRAM void EnvLight_Sample_L(const SData_Light& data,LMD& lmd,uint& seed){
	//RT_CALLABLE_PROGRAM Irradiance4 EnvLight_Sample_L(const SData_Light& data,const Pos& pos_g,Dir& wi_g,float& pdf,uint& seed){
	RandomFloat2 rf2=CF_GETSAMPLE2(seed);
	UV2 uv; float mapPDF;
	CF_DISTRIBUTION2D_SAMPLECONTINUOUS(data,rf2,uv,mapPDF);
	//dtrb2D.SampleDiscrete(ls,uv,mapPDF);

	//rtPrintf("rt2[%f,%f],PDF[%f],,",rf2.x,rf2.y,mapPDF);
#if 0
	//Testing
	PDF=1.0f;
	wi_g=make_float3(0.f,1.f,0.f);
	return make_float3(mapPDF);
#endif

	if(mapPDF==0.f){
		lmd.L=make_float4(0.f);
		return;
	}

	float sintheta;
	UVToNor(uv,lmd.wi_g,sintheta);

#if 0
			if(uint(lmd.dim.x*.5f)==lmd.index.x && uint(lmd.dim.y*.5f)==lmd.index.y){
				rtPrintf("sintheta[%f],,",sintheta);
			}
#endif

	if(sintheta<=0.f){
		lmd.L=make_float4(0.f);
	} else{
#if 0
		rtPrintf("wi_g is:%f,%f,%f\n",wi_g.x,wi_g.y,wi_g.z);
#endif
		CalcWithMatrix(lmd.wi_g,data.bufferID_worldMatrix);
		lmd.wi_g*=M_MAX_FLOAT;
		lmd.PDF=mapPDF/(M_2PIf*M_PIf*sintheta);
		lmd.L=data.HDR_Intensity*rtTex2D<float4>(data.HDR_textureSamplerID,uv.x,uv.y); //test:Radiance4 L=make_float4(0.1f);;
	}
}

RT_CALLABLE_PROGRAM float3 EnvLight_Power(){ return make_float3(0.f); }

RT_CALLABLE_PROGRAM float EnvLight_Pdf(const SData_Light& data,const Pos&,const Dir &wi){
	UV2 uv;
	DirToUV(wi,uv);
	const float sintheta=sinf(uv.y*M_PIf);
	float tmp=CF_DISTRIBUTION2D_PDF(data,uv);
	return tmp/(M_2PIf*M_PIf*sintheta);
}
/**
* 针对未与场景几何体对象碰撞的光线，无穷远区域光源须向其“贡献”辐射度。对此，可向Light基类加入一方法，沿未碰撞光线并根据当前光源返回发射辐射度。
* 针对其他类型光源，默认实现并不返回辐射度。
*/
//RT_FUNC float3 Le(const RayDifferential &r) const;
//RT_FUNC void SHProject(const Pos &p,float pEpsilon,int lmax,float3 *coeffs){ }
#pragma endregion

#pragma endregion
