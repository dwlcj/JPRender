#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu\optixu_matrix_namespace.h>
#include "cu_common.h"
#include "intersection.h"
#include "../../host_device/data_context.h"
#include "../../host_device/data_renderer.h"

using namespace optix;

rtDeclareVariable(Intersection,aInset,attribute aInset,);
rtDeclareVariable(rtObject,V_C_TOP_OBJECT,,);
rtDeclareVariable(SData_Context,V_C_DATA_CONTEXT,,);
rtDeclareVariable(SData_Renderer,V_C_DATA_RENDERER,,);
rtDeclareVariable(int,prdP,rtPayload,);

////////////////////////////////////////////////////////////////////////////////////////////////////
// launching program
////////////////////////////////////////////////////////////////////////////////////////////////////

RT_PROGRAM void PgmLaunch_pick(){
  /// 1. ��ȡC���ύ���������е����ݣ�
  float4 data=GET_F4_1D(V_C_DATA_CONTEXT.picker_bufID,0);
  const Nor rayDir=make_float3(data.x,data.y,data.z);

  /// 2. �������߲����䣻
  Ray ray=make_Ray(V_C_DATA_RENDERER.camera_pos,rayDir,OptixRayTypes::PICKUP,V_C_DATA_RENDERER.ray_clamp_min,V_C_DATA_RENDERER.ray_clamp_max);
  int w=-1;
  rtTrace(V_C_TOP_OBJECT,ray,w);

  /// 3. �����ȡ����meshID������������C��ʹ�ã�
	data.w=(float)w;
  SET_F4_1D(V_C_DATA_CONTEXT.picker_bufID,0,data);
}

RT_PROGRAM void PgmCH_pick(){
  prdP=aInset.nodeIndex;
}
