#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu\optixu_matrix_namespace.h>
#include "cu_common.h"
#include "../../host_device/data_context.h"
#include "../../host_device/data_renderer.h"

using namespace optix;

struct SData_prd_depthRendering{
  float depth;
};

rtDeclareVariable(uint2,dLd,rtLaunchDim,);
rtDeclareVariable(uint2,dLi,rtLaunchIndex,);
rtDeclareVariable(float,dT,rtIntersectionDistance,);
rtDeclareVariable(rtObject,V_C_TOP_OBJECT,,);
rtDeclareVariable(SData_Renderer,V_C_DATA_RENDERER,,);
rtDeclareVariable(SData_prd_depthRendering,prd_depthRendering,rtPayload,);

//rtDeclareVariable(rtCallableProgramId<RandomFloat1(uint&)>,CF_GETSAMPLE1,,);
rtDeclareVariable(rtCallableProgramId<uint(uint,uint)>,CF_GETRANDOMSEED,,);
rtDeclareVariable(rtCallableProgramId<RandomFloat2(uint&)>,CF_GETSAMPLE2,,);

#define __ONLY_FUNC__
#include "prepare_launch.h"
#undef __ONLY_FUNC__

RT_PROGRAM void PgmLaunch_depth(){
	Nor direction; uint seed;
	PrepareLaunch(direction,seed);

#if 0
		if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
			rtPrintf("directions[%f,%f,%f]\n",direction.x,direction.y,direction.z);
		}
#endif

  SData_prd_depthRendering prd;
  prd.depth=M_MAX_FLOAT;
  Ray ray=make_Ray(V_C_DATA_RENDERER.camera_pos,direction,OptixRayTypes::DEPTH,V_C_DATA_RENDERER.ray_clamp_min,V_C_DATA_RENDERER.ray_clamp_max);
  rtTrace(V_C_TOP_OBJECT,ray,prd);


  Radiance4 L=make_float4(0.f);
  const float delta=V_C_DATA_RENDERER.DepthRenderer_depth_max-prd.depth;
  if(delta>0.f){
    float pct=delta/(V_C_DATA_RENDERER.DepthRenderer_depth_max-V_C_DATA_RENDERER.DepthRenderer_depth_min);
		L=make_float4(pct);

		if(V_C_DATA_RENDERER.frame_current>1u){
			const float weight=1.f/(float)V_C_DATA_RENDERER.frame_current;
			Radiance4 tmp=GET_F4_2D(V_C_DATA_RENDERER.output_launchBufID,dLi);
			L=lerp(tmp,L,weight);
		}
  }
	L.w=1.f;

	SET_F4_2D(V_C_DATA_RENDERER.output_launchBufID,dLi,L);
}

RT_PROGRAM void PgmCH_depth(){

  prd_depthRendering.depth=dT;

#if 0
  if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
    rtPrintf("inv_depth:%f\n",prd_depthRendering.depth);
  }
  return;
#endif
}