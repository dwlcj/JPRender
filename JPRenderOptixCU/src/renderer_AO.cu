#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu\optixu_matrix_namespace.h>
#include "cu_common.h"
#include "intersection.h"
#include "per_ray_data.h"
#include "../../host_device/data_context.h"
#include "../../host_device/data_renderer.h"
#include "../../host_device/data_instance.h"

using namespace optix;

#include "light_model_data.h"

rtDeclareVariable(uint2,dLd,rtLaunchDim,);
rtDeclareVariable(uint2,dLi,rtLaunchIndex,);
rtDeclareVariable(Ray,dRay,rtCurrentRay,);
rtDeclareVariable(float,dT,rtIntersectionDistance,);
rtDeclareVariable(Intersection,aInset,attribute aInset,);
rtDeclareVariable(rtObject,V_C_TOP_OBJECT,,);
rtDeclareVariable(SData_Context,V_C_DATA_CONTEXT,,);
rtDeclareVariable(SData_Renderer,V_C_DATA_RENDERER,,);
rtDeclareVariable(SData_Instance,V_C_DATA_INSTANCE,,);
rtDeclareVariable(PRD,prdR,rtPayload,);


// callable
rtDeclareVariable(rtCallableProgramId<uint(uint,uint)>,CF_GETRANDOMSEED,,);
rtDeclareVariable(rtCallableProgramId<RandomFloat1(uint&)>,CF_GETSAMPLE1,,);
rtDeclareVariable(rtCallableProgramId<RandomFloat2(uint&)>,CF_GETSAMPLE2,,);
rtDeclareVariable(rtCallableProgramId<Dir(const RandomFloat2&,float,float)>,CF_COSINESAMPLEHEMISPHERE_QUICKER,,);
rtDeclareVariable(rtCallableProgramId<Nor(const SData_Texture&,Intersection)>,CF_GETTEXTURE_NORMAL,,);
rtBuffer<rtCallableProgramId<void(LMD&,uint&)>,1> buf_lm_sample_F;

#define __ONLY_FUNC__
#include "common_AO_launch.h"
#undef __ONLY_FUNC__

RT_PROGRAM void PgmLaunch_AO(){

	/// 1. ����������������ȡ���ӣ�
	const int index=dLi.y*dLd.x+dLi.x;
	uint seed=CF_GETRANDOMSEED(index,V_C_DATA_RENDERER.frame_current);


	Radiance4 fRadiance=make_float4(AOCommonLaunch(seed));

	if(V_C_DATA_RENDERER.frame_current>1u){
		float weight=1.f/(float)V_C_DATA_RENDERER.frame_current;
		Radiance4 L=GET_F4_2D(V_C_DATA_RENDERER.output_launchBufID,dLi);
		fRadiance=lerp(L,fRadiance,weight);
		fRadiance.w=1.f;
	} 
	SET_F4_2D(V_C_DATA_RENDERER.output_launchBufID,dLi,fRadiance);

}


RT_PROGRAM void PgmCH_AO(){
	SData_Material& md=
		rtBufferId<SData_Material,1>(V_C_DATA_INSTANCE.materialBufferID)[V_C_DATA_INSTANCE.materialBufferOffset];
	prdR.hp_g=dRay.origin+dT*dRay.direction;
	if(md.materialIndex==MATERIAL_GLASS_INDEX){
		prdR.AO_nextIteration=true;


		LMD lmd;
		lmd.wo_g=-dRay.direction;
		lmd.hitPos_g=prdR.hp_g;
		lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,aInset.geoNor));
		uint seed=prdR.seed;

		const SData_Texture& data=md.GE_texture;
		Nor nor=CF_GETTEXTURE_NORMAL(data,aInset);
		lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,nor));
		lmd.IOR=md.BS_ior;
		const RandomFloat1 u=CF_GETSAMPLE1(seed);
		float fr=fresnel_schlick(fabsf(dot(lmd.wo_g,lmd.sn_g)),md.BS_fresnel.exponent,md.BS_fresnel.fmin,md.BS_fresnel.fmax);
		if(u<fr){
			buf_lm_sample_F[CFINDEX_LM_SPECULAR_REFLECT](lmd,seed);
		} else{
			buf_lm_sample_F[CFINDEX_LM_SPECULAR_REFRACT](lmd,seed);
		}
		prdR.wi_g=lmd.wi_g;
		prdR.seed=seed;

	} else{
		prdR.AO_intensity=md.BS_AOIntensity;
		if(prdR.uDepth==0){
			// AO
			Dir n_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,aInset.geoNor));
			//n_g=faceforward(n_g,-dRay.direction,n_g); // ��ת���׳��ֺڱߣ�
			prdR.n_g=normalize(n_g);
			++prdR.uDepth;
		} else{
			// result based AO;

			prdR.AO_distancePercent=dT/V_C_DATA_RENDERER.ray_clamp_max_AO;
			/*
			float distanceAttenuation=clamp(dT/V_C_DATA_RENDERER.ray_clamp_max_AO,0.f,1.f);
			float areaAttenuation=V_C_DATA_RENDERER.ray_clamp_min_AO;
			prdR.AO_distancePercent=(distanceAttenuation+areaAttenuation)*.5f;
			*/
		}
	}
}

