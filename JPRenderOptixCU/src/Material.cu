#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu\optixu_matrix_namespace.h>
#include "cu_common.h"
#include "intersection.h"
#include "per_ray_data.h"
#include "../../host_device/data_renderer.h"
#include "../../host_device/data_light.h"
#include "../../host_device/data_instance.h"

using namespace optix;

#include "light_model_data.h"

struct PRD_Shadow{
	bool canSeeLight;
};

rtDeclareVariable(Ray,dRay,rtCurrentRay,);
rtDeclareVariable(float,dT,rtIntersectionDistance,);
rtDeclareVariable(PRD,prdR,rtPayload,);
rtDeclareVariable(PRD_Shadow,prdS,rtPayload,);
rtDeclareVariable(Intersection,aInset,attribute aInset,);
rtDeclareVariable(SData_Instance,V_C_DATA_INSTANCE,,);
rtDeclareVariable(SData_Renderer,V_C_DATA_RENDERER,,);
rtDeclareVariable(rtObject,V_C_TOP_OBJECT,,);

rtDeclareVariable(rtCallableProgramId<RandomFloat1(uint&)>,CF_GETSAMPLE1,,);
rtDeclareVariable(rtCallableProgramId<RandomFloat2(uint&)>,CF_GETSAMPLE2,,);
rtDeclareVariable(rtCallableProgramId<Color4(int TSID,const Dir&)>,CF_GETSKYBOXRADIANCE,,);
rtDeclareVariable(rtCallableProgramId<float(int,int,const Nor&,float,Color4&)>,CF_GETENVRADIANCE,,);
rtDeclareVariable(rtCallableProgramId<float(float3,float)>,CF_CHECKERBOARD3,,);
rtDeclareVariable(rtCallableProgramId<Color4(int,int,UV2)>,CF_GETTEXTURE_DIFFUSE,,);
rtDeclareVariable(rtCallableProgramId<Color4(int,int,UV2)>,CF_GETTEXTURE_GLOSSY,,);
rtDeclareVariable(rtCallableProgramId<Nor(const SData_Texture&,Intersection)>,CF_GETTEXTURE_NORMAL,,);
rtDeclareVariable(rtCallableProgramId<Nor(int,const Pos&,const Nor&)>,CF_GETTRIPLANENORMAL,,);
rtDeclareVariable(rtCallableProgramId<uint(uint,uint)>,CF_GETRANDOMSEED,,);

// light;
rtBuffer<rtCallableProgramId<float(const SData_Light&,const Pos&,const Dir&)>,1> buf_light_PDF;
rtBuffer<rtCallableProgramId<Radiance4(const SData_Light&)>,1> buf_light_Power;
rtBuffer<rtCallableProgramId<void(const SData_Light&,LMD&,uint&)>,1> buf_light_sample_L;

// light model;
//rtBuffer<rtCallableProgramId<void(LMD&,uint&)>,1> buf_lm_PDF;
rtBuffer<rtCallableProgramId<void(LMD&,uint&)>,1> buf_lm_F;
rtBuffer<rtCallableProgramId<void(LMD&,uint&)>,1> buf_lm_sample_F;
//rtBuffer<rtCallableProgramId<void(LMD&)>,1> buf_lm_albedo;


#define __ONLY_FUNC__
#include "common_texture_operation.h"
#undef __ONLY_FUNC__


#pragma region helper
RT_FUNC_S Color4 MergeColor(Color4 source,Color4 target,float gamma){
	Color4 tmp=source*source.w+(1-source.w)*target;
	const float F{1.f/gamma};
	tmp.x=powf(tmp.x,F);
	tmp.y=powf(tmp.y,F);
	tmp.z=powf(tmp.z,F);
	tmp.w=1.f;
	return tmp;
}

RT_FUNC_F float luminanceCIE(const float4& rgba){
	const float4 ntsc_luminance={0.30f, 0.59f, 0.11f,0.f};
	return dot(rgba,ntsc_luminance);
}

RT_FUNC_S void HandelLight(const uint BRDFindex,LMD& lmd,uint& seed){
#if 0
	//test:
	return make_float4(1.f);
#endif

	Radiance4 L_out=make_float4(0.f);
	Color4 C_out=make_float4(1.f);

	int N=rtBufferId<SData_Light,1>(V_C_DATA_RENDERER.bufferID_lights).size();
	for(uint i=0u; i<N; ++i){
		const SData_Light& ld=rtBufferId<SData_Light,1>(V_C_DATA_RENDERER.bufferID_lights)[i];

		buf_light_sample_L[ld.lightFuncIndex](ld,lmd,seed);
		if(luminanceCIE(lmd.L)<=M_MIN_RADIANCE)continue;
		if(lmd.PDF<=M_MIN_PDF)continue;
		const float dis=length(lmd.wi_g);
		lmd.wi_g=lmd.wi_g/dis;

		const float cosTheta=lmd.WiDotN();
		if(cosTheta>0.f){
			Ray ray=make_Ray(lmd.hitPos_g,lmd.wi_g,OptixRayTypes::SHADOW,V_C_DATA_RENDERER.ray_clamp_min,dis);
			PRD_Shadow prd;
			prd.canSeeLight=true;
			rtTrace(V_C_TOP_OBJECT,ray,prd);

			if(prd.canSeeLight){
				buf_lm_F[BRDFindex](lmd,seed);
				L_out+=lmd.BRDF*lmd.L*cosTheta/lmd.PDF;
			}
		}
	}// end for
#if 0
	if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
		rtPrintf("C_out[%f,%f,%f],,",C_out.x,C_out.y,C_out.z);
	}
#endif
	lmd.L=L_out;
}

RT_FUNC_S void Miss(){
#if 0
	//test
	if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
		rtPrintf("missed - in PgmMiss");
	}
#endif
	Nor rayDir=dRay.direction;
	if(prdR.uDepth==0u){
		if(V_C_DATA_RENDERER.skybox_enableBackground){
			prdR.color_out=CF_GETSKYBOXRADIANCE(V_C_DATA_RENDERER.skybox_TSID_show,rayDir);
			prdR.luminance_out=make_float4(1.f);
		} else{
			prdR.luminance_out=make_float4(0.f);
		}

		if(V_C_DATA_RENDERER.enableAIdenoise){
			prdR.albedo=make_float4(0.f,0.f,0.f,1.f);
			prdR.n_g=make_float3(0.f);
		}
	} else{
		Color4 color;
		prdR.luminance_out=make_float4(CF_GETENVRADIANCE(V_C_DATA_RENDERER.skybox_TSID_show,V_C_DATA_RENDERER.bufferID_lights,rayDir,prdR.lastMaterialEnvBlend,color));
		prdR.color_out=color;
	}

	// ��ֹ���ߣ�ʹ�䲻Ҫ����������
	prdR.missed=true;
}

RT_FUNC_F void AdjustVecBySize(const float3& f3,const float size,RandomFloat2* rf2){
	float3 d,d2;
	const float INV_SIZE=1.f/size;
	float3 v1=make_float3(1.4142f,-1.4142f,0.f);
	float3 v2=make_float3(0,1.4142f,-1.4142f);
	float3 v3=make_float3(1.4142f,0.f,-1.4142f);
	d.x=int(floor(dot(v1,f3)*INV_SIZE));
	d.y=int(floor(dot(v2,f3)*INV_SIZE));
	d.z=int(floor(dot(v3,f3)*INV_SIZE));
	d*=size;

	v1=make_float3(0.935f,0.25f,0.25f);
	v2=make_float3(0.25f,0.935f,0.25f);
	v3=make_float3(0.25f,0.25f,0.935f);
	d2.x=int(floor(dot(v1,f3)*INV_SIZE));
	d2.y=int(floor(dot(v2,f3)*INV_SIZE));
	d2.z=int(floor(dot(v3,f3)*INV_SIZE));
	d2*=size;

	float a=sin(d.x*12.9898f+d.y*78.233f+d.z*45.164f+d2.x*94.673f+d2.y*45.164f+d2.z*78.233f);
	a=fabsf(a*43758.5453f);
	rf2->x=a-int(a);

	a=sin(d.y*12.9898f+d.z*78.233f+d2.x*45.164f+d2.y*94.673f+d2.z*45.164f+d.x*78.233f);
	a=fabsf(a*43758.5453f);
	rf2->y=a-int(a);

}

RT_FUNC_S void HandleMetallicFlakeAndBase(LMD& lmd,uint& seed,const Color4& color_diffuse){
	//uint index=uint(length(lmd.hitPos_g)*0xFFFFFF)%0xFFFFFFFF;
	//uint newSeed=CF_GETRANDOMSEED(aInset.texCoord.x,11);
	SData_Material& md=rtBufferId<SData_Material,1>(V_C_DATA_INSTANCE.materialBufferID)[V_C_DATA_INSTANCE.materialBufferOffset];
	RandomFloat2 v2;
	AdjustVecBySize(lmd.hitPos_g,md.CF_size,&v2);
	if(v2.x<md.CF_density){
		//flake;
		v2.x/=md.CF_density;
		const float theta=atanf(md.CF_irregularity*sqrtf(-logf(1.f-v2.x)));
		const float sintheta=sinf(theta);
		const float costheta=cosf(theta);
		//const float phi=M_PI_2f;
		const float phi=M_2PIf*v2.y;

		Onb onb(lmd.sn_g);
		Nor flakeNor=make_float3(sintheta * cosf(phi),sintheta * sinf(phi),costheta);
		onb.inverse_transform(flakeNor);
		flakeNor=normalize(flakeNor);

		//buf_lm_sample_F[CFINDEX_LM_SPECULAR_REFLECT](lmd,seed);
		//change to :
		lmd.wi_g=reflect(-lmd.wo_g,flakeNor);
		if(lmd.WiDotN()<0.f){
			lmd.wi_g*=-1.f;
		}
		lmd.NormalizeWi();
		lmd.PDF=1.f;
		lmd.BRDF=1.f/fabsf(lmd.WiDotN());


		//lmd.sn_g=oldSn_g;
		prdR.color_out=md.CF_color;
		if(prdR.lastMaterialEnvBlend>1.f) prdR.lastMaterialEnvBlend=1.f;
		lmd.L=make_float4(0.f);
	} else{
		//base;
		HandelLight(CFINDEX_LM_LAMBERTIAN,lmd,seed);
		buf_lm_sample_F[CFINDEX_LM_LAMBERTIAN](lmd,seed);
		prdR.color_out=color_diffuse;
		prdR.lastMaterialEnvBlend=0.f;
		if(prdR.uDepth<V_C_DATA_RENDERER.ray_iterateDepth-1u){
			prdR.uDepth=V_C_DATA_RENDERER.ray_iterateDepth-1u;
		}
	}
}
#pragma endregion



RT_PROGRAM void PgmAH_radiance_shadow(){
	prdS.canSeeLight=false;
	rtTerminateRay();
}




RT_PROGRAM void PgmCH_radiance(){
#if 0
	//test
	if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
		rtPrintf("closet hit - \n");
	}
#endif


	/// 2. basic data;
	LMD lmd;
	lmd.wo_g=-dRay.direction;
	lmd.hitPos_g=dRay.origin+dT*dRay.direction;
	lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,aInset.geoNor));
	uint seed=prdR.seed;
	const SData_Material& md=
		rtBufferId<SData_Material,1>(V_C_DATA_INSTANCE.materialBufferID)[V_C_DATA_INSTANCE.materialBufferOffset];


	/// 3. material color;
	//Color4 color_diffuse=CF_GETTEXTURE_DIFFUSE(md.BS_textureDiffuse,aInset.texCoord);
	Color4 color_diffuse=CF_GETTEXTURE_DIFFUSE(V_C_DATA_INSTANCE.materialBufferID
		,V_C_DATA_INSTANCE.materialBufferOffset,aInset.texCoord);
	Color4 color_glossy=CF_GETTEXTURE_GLOSSY(V_C_DATA_INSTANCE.materialBufferID
		,V_C_DATA_INSTANCE.materialBufferOffset,aInset.texCoord);

#if 0
	//test
	rtPrintf("color_diffuse,[%f,%f,%f]\n",color_diffuse.x,color_diffuse.y,color_diffuse.z);
#endif

	/// 4. BRDF;
	switch(md.materialIndex){
	case MATERIAL_EMISSIVE_INDEX:
	{
		if(prdR.uDepth==0u){
			prdR.color_out=color_diffuse*md.BS_AOIntensity;
			prdR.luminance_out=make_float4(1.f);
		} else{
			prdR.luminance_out=make_float4(0.f);
		}
		prdR.missed=true;
		return;
	}
	break;
	case MATERIAL_SHADOW_INDEX:
	{
		Miss();
		int N=rtBufferId<SData_Light,1>(V_C_DATA_RENDERER.bufferID_lights).size();
		for(uint i=0u; i<N; ++i){
			const SData_Light& ld=rtBufferId<SData_Light,1>(V_C_DATA_RENDERER.bufferID_lights)[i];

			buf_light_sample_L[ld.lightFuncIndex](ld,lmd,seed);
			const float dis=length(lmd.wi_g);
			lmd.wi_g=lmd.wi_g/dis;

			Ray ray=make_Ray(lmd.hitPos_g,lmd.wi_g,OptixRayTypes::SHADOW,V_C_DATA_RENDERER.ray_clamp_min,dis);
			PRD_Shadow prd;
			prd.canSeeLight=true;
			rtTrace(V_C_TOP_OBJECT,ray,prd);

			if(!prd.canSeeLight){
				prdR.luminance_out*=md.GE_shadowAttenuation;
			}
		}// end for
		prdR.seed=seed;
		return;
	}
	break;
	case MATERIAL_GLASS_INDEX:
	{
		const SData_Texture& data=md.GE_texture;
		Nor nor=CF_GETTEXTURE_NORMAL(data,aInset);
		lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,nor));
		lmd.IOR=md.BS_ior;
		const RandomFloat1 u=CF_GETSAMPLE1(seed);
		float fr=fresnel_schlick(fabsf(dot(lmd.wo_g,lmd.sn_g)),md.BS_fresnel.exponent,md.BS_fresnel.fmin,md.BS_fresnel.fmax);
		if(u<fr){
			buf_lm_sample_F[CFINDEX_LM_SPECULAR_REFLECT](lmd,seed);
			prdR.color_out=color_glossy;
		} else{
			buf_lm_sample_F[CFINDEX_LM_SPECULAR_REFRACT](lmd,seed);
			prdR.color_out=color_diffuse;
		}
		lmd.L=make_float4(0.f);
		if(prdR.lastMaterialEnvBlend>1.f) prdR.lastMaterialEnvBlend=1.f;
	}
	break;
	case MATERIAL_METAL_INDEX:
	{
		const SData_Texture& data=md.GE_texture;
		Nor nor=CF_GETTEXTURE_NORMAL(data,aInset);
		lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,nor));
		lmd.roughness=1.f/md.BS_roughness;
		HandelLight(CFINDEX_LM_TORRANCESPARROW,lmd,seed);
		prdR.color_out=color_diffuse*color_glossy;
		buf_lm_sample_F[CFINDEX_LM_TORRANCESPARROW](lmd,seed);
		const float blend=1.f-md.BS_roughness;
		if(prdR.lastMaterialEnvBlend>blend) prdR.lastMaterialEnvBlend=blend;
		if(prdR.uDepth<V_C_DATA_RENDERER.ray_iterateDepth-1u){
			prdR.uDepth=V_C_DATA_RENDERER.ray_iterateDepth-1u;
		}
	}
	break;
	case MATERIAL_METAL_BRUSHED_INDEX:
	{
		const SData_Texture& data=md.GE_texture;
		Nor nor=CF_GETTEXTURE_NORMAL(data,aInset);
		lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,nor));
		Pos center_g=normalize(rtTransformPoint(RT_OBJECT_TO_WORLD,make_float3(md.BS_center)));

		Dir to_center_g=center_g-lmd.hitPos_g;
		////////////////////////////////////////////////////////////////////////////////////////////////////
		//TODO: fix in the future;
		//if(data.M_MPROP_BS_usePlanar) tangent=::normalize(::cross(n_g,center-hp_g));
		//else tangent=::normalize(::cross(n_g,center));
		// replace the following with above;
		lmd.tangent_g=normalize(cross(lmd.sn_g,to_center_g));
		////////////////////////////////////////////////////////////////////////////////////////////////////
		lmd.bitangent_g=normalize(cross(lmd.sn_g,lmd.tangent_g));
		lmd.reflatance.x=md.BS_reflectanceDiffuse;
		lmd.reflatance.y=md.BS_reflectanceSpecular;
		lmd.anisotropicUV.x=md.BS_anisotropyU;
		lmd.anisotropicUV.y=md.BS_anisotropyV;
		HandelLight(CFINDEX_LM_ASHIKHMIN_SHIRLEY,lmd,seed);
		prdR.color_out=color_diffuse*color_glossy;
		buf_lm_sample_F[CFINDEX_LM_ASHIKHMIN_SHIRLEY](lmd,seed);
		if(prdR.lastMaterialEnvBlend>1.f) prdR.lastMaterialEnvBlend=1.f;
		if(prdR.uDepth<V_C_DATA_RENDERER.ray_iterateDepth-1u){
			prdR.uDepth=V_C_DATA_RENDERER.ray_iterateDepth-1u;
		}
	}
	break;
	case MATERIAL_WOOD_INDEX:
	{
		const RandomFloat1 u=CF_GETSAMPLE1(seed);
		if(u<md.CC_weight){
			// coat
			const SData_Texture& data=md.CC_texture;
			Nor nor=CF_GETTEXTURE_NORMAL(data,aInset);
			lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,nor));
			buf_lm_sample_F[CFINDEX_LM_SPECULAR_REFLECT](lmd,seed);
			lmd.L=make_float4(0.f);
			prdR.color_out=md.CC_texture.color;
			if(prdR.lastMaterialEnvBlend>1.f) prdR.lastMaterialEnvBlend=1.f;
		} else{
			const SData_Texture& data=md.GE_texture;
			Nor nor=CF_GETTEXTURE_NORMAL(data,aInset);
			lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,nor));
			HandelLight(CFINDEX_LM_LAMBERTIAN,lmd,seed);
			prdR.color_out=color_diffuse;
			buf_lm_sample_F[CFINDEX_LM_LAMBERTIAN](lmd,seed);
			prdR.lastMaterialEnvBlend=0.f;
			prdR.uDepth=V_C_DATA_RENDERER.ray_iterateDepth;
		}
	}
	break;
	case MATERIAL_RUBBER_INDEX:
	{
		const SData_Texture& data=md.GE_texture;
		Nor nor=CF_GETTEXTURE_NORMAL(data,aInset);
		lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,nor));
		float fr;
		if(md.BS_fresnel.fmax<md.BS_fresnel.fmin){
			fr=fresnel_schlick(fabsf(lmd.WoDotN()),md.BS_fresnel.exponent,md.BS_fresnel.fmax,md.BS_fresnel.fmin);
			fr=1.f-fr;
		} else{
			fr=fresnel_schlick(fabsf(lmd.WoDotN()),md.BS_fresnel.exponent,md.BS_fresnel.fmin,md.BS_fresnel.fmax);
		}

		const RandomFloat1 u1=CF_GETSAMPLE1(seed);
		if(u1<fr){
			buf_lm_sample_F[CFINDEX_LM_SPECULAR_REFLECT](lmd,seed);
			prdR.color_out=md.BS_colorReflect;
			if(prdR.lastMaterialEnvBlend>1.f) prdR.lastMaterialEnvBlend=1.f;
			lmd.L=make_float4(0.f);
		} else{
			HandelLight(CFINDEX_LM_LAMBERTIAN,lmd,seed);
			prdR.color_out=color_diffuse*color_glossy;
			buf_lm_sample_F[CFINDEX_LM_LAMBERTIAN](lmd,seed);
			prdR.lastMaterialEnvBlend=0.f;
			prdR.uDepth=V_C_DATA_RENDERER.ray_iterateDepth;
		}
	}
	break;
	case MATERIAL_LEATHER_INDEX:
	{
		GetNormal_triplanar(md.GE_texture,lmd.hitPos_g,lmd.sn_g);
		float fr;
		if(md.BS_fresnel.fmax<md.BS_fresnel.fmin){
			fr=fresnel_schlick(fabsf(lmd.WoDotN()),md.BS_fresnel.exponent,md.BS_fresnel.fmax,md.BS_fresnel.fmin);
			fr=1.f-fr;
		} else{
			fr=fresnel_schlick(fabsf(lmd.WoDotN()),md.BS_fresnel.exponent,md.BS_fresnel.fmin,md.BS_fresnel.fmax);
		}

		const RandomFloat1 u1=CF_GETSAMPLE1(seed);
		if(u1<fr){
			buf_lm_sample_F[CFINDEX_LM_SPECULAR_REFLECT](lmd,seed);
			prdR.color_out=md.BS_colorReflect;
			if(prdR.lastMaterialEnvBlend>1.f) prdR.lastMaterialEnvBlend=1.f;
			lmd.L=make_float4(0.f);
		} else{
			HandelLight(CFINDEX_LM_LAMBERTIAN,lmd,seed);
			prdR.color_out=color_diffuse*color_glossy;
			buf_lm_sample_F[CFINDEX_LM_LAMBERTIAN](lmd,seed);
			prdR.lastMaterialEnvBlend=0.f;
			prdR.uDepth=V_C_DATA_RENDERER.ray_iterateDepth;
		}
	}
	break;
	case MATERIAL_CARPAINT_UNIQUE_CLEAR_INDEX:
	{
		const RandomFloat2 u2=CF_GETSAMPLE2(seed);
		if(u2.x<md.CC_weight){

			float fr=fresnel_schlick(fabsf(lmd.WoDotN()),md.CC_fresnel.exponent,md.CC_fresnel.fmin,md.CC_fresnel.fmax);
			if(u2.y<fr){
				// coat
				const SData_Texture& data=md.CC_texture;
				Nor nor=CF_GETTEXTURE_NORMAL(data,aInset);
				lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,nor));
				buf_lm_sample_F[CFINDEX_LM_SPECULAR_REFLECT](lmd,seed);
				prdR.color_out=md.CC_texture.color;
				if(prdR.lastMaterialEnvBlend>1.f) prdR.lastMaterialEnvBlend=1.f;
				lmd.L=make_float4(0.f);
			} else{
				HandelLight(CFINDEX_LM_LAMBERTIAN,lmd,seed);
				prdR.color_out=color_diffuse;
				buf_lm_sample_F[CFINDEX_LM_LAMBERTIAN](lmd,seed);
				prdR.lastMaterialEnvBlend=0.f;
				if(prdR.uDepth<V_C_DATA_RENDERER.ray_iterateDepth-1u){
					prdR.uDepth=V_C_DATA_RENDERER.ray_iterateDepth-1u;
				}
			}

		} else{
			HandelLight(CFINDEX_LM_LAMBERTIAN,lmd,seed);
			prdR.color_out=color_diffuse;
			buf_lm_sample_F[CFINDEX_LM_LAMBERTIAN](lmd,seed);
			prdR.lastMaterialEnvBlend=0.f;
			if(prdR.uDepth<V_C_DATA_RENDERER.ray_iterateDepth-1u){
				prdR.uDepth=V_C_DATA_RENDERER.ray_iterateDepth-1u;
			}
		}
	}
	break;
	case MATERIAL_CARPAINT_METALLIC_INDEX:
	{

		const RandomFloat2 u2=CF_GETSAMPLE2(seed);
		if(u2.x<md.CC_weight){

			float fr=fresnel_schlick(fabsf(lmd.WoDotN()),md.CC_fresnel.exponent,md.CC_fresnel.fmin,md.CC_fresnel.fmax);
			if(u2.y<fr){
				// coat
				const SData_Texture& data=md.CC_texture;
				Nor nor=CF_GETTEXTURE_NORMAL(data,aInset);
				lmd.sn_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,nor));
				buf_lm_sample_F[CFINDEX_LM_SPECULAR_REFLECT](lmd,seed);
				prdR.color_out=md.CC_texture.color;
				if(prdR.lastMaterialEnvBlend>1.f) prdR.lastMaterialEnvBlend=1.f;
				lmd.L=make_float4(0.f);
			} else{
				HandleMetallicFlakeAndBase(lmd,seed,color_diffuse);
			}

		} else{
			HandleMetallicFlakeAndBase(lmd,seed,color_diffuse);
			}
		}
	break;
	case MATERIAL_CARPAINT_UNIQUE_INDEX:
	case MATERIAL_MATTE_INDEX:
	{
		HandelLight(CFINDEX_LM_LAMBERTIAN,lmd,seed);
		prdR.color_out=color_diffuse;
		buf_lm_sample_F[CFINDEX_LM_LAMBERTIAN](lmd,seed);
		prdR.lastMaterialEnvBlend=0.f;
		prdR.uDepth=V_C_DATA_RENDERER.ray_iterateDepth;
#if 0
		//test
		rtPrintf("bufid[%d],bufoffset[%d]\n",V_C_DATA_INSTANCE.materialBufferID,V_C_DATA_INSTANCE.materialBufferOffset);
		rtPrintf("dc[%f]\n",md.BS_textureDiffuse.color.x);
#endif
	}
	break;
   case MATERIAL_UV_INDEX:
   {
     HandelLight(CFINDEX_LM_LAMBERTIAN,lmd,seed);
     prdR.color_out=color_diffuse;
     buf_lm_sample_F[CFINDEX_LM_LAMBERTIAN](lmd,seed);
     prdR.lastMaterialEnvBlend=0.f;
     prdR.uDepth=V_C_DATA_RENDERER.ray_iterateDepth;
   }
   break;
	case MATERIAL_DEFAULT_INDEX:
	default:
	{
		prdR.color_out=color_diffuse*(.05f+.95f*CF_CHECKERBOARD3(lmd.hitPos_g,md.BS_checkboardSize));
		prdR.luminance_out=make_float4(1.f);
		prdR.missed=true;
		return;
	}
	break;
	}

	/// 5. return;
	prdR.materialIndex=md.materialIndex;
	prdR.luminance_out=lmd.L;
	prdR.BRDF=lmd.BRDF;
	prdR.PDF=lmd.PDF;
	prdR.seed=seed;
	prdR.hp_g=lmd.hitPos_g;
	prdR.n_g=lmd.sn_g;
	prdR.wi_g=lmd.wi_g;
	if(V_C_DATA_RENDERER.enableAIdenoise) prdR.albedo=prdR.color_out;
}




RT_PROGRAM void PgmMiss_radiance(){
	Miss();
}

#if 0
//test
if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
	//rtPrintf("prd data is:uv[%f,%f],wo_g[%f,%f,%f],hp_g[%f,%f,%f]",prdR.uv.x,prdR.uv.y,prdR.wo_g.x,prdR.wo_g.y,prdR.wo_g.z,prdR.hp_g.x,prdR.hp_g.y,prdR.hp_g.z);
	//rtPrintf("wi_g[%f,%f,%f]",prdR.wi_g.x,prdR.wi_g.y,prdR.wi_g.z);
	rtPrintf("gn_g[%f,%f,%f],depth[%d],BRDF[%d]\n",prdR.gn_g.x,prdR.gn_g.y,prdR.gn_g.z,prdR.uDepth,prdR.BRDFindex);
}
#endif
