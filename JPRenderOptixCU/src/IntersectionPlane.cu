#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu\optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "cu_common.h"
#include "intersection.h"

using namespace optix;

/*
rtDeclareVariable(float3,plane,,);
rtDeclareVariable(float3,v1,,);
rtDeclareVariable(float3,v2,,);
rtDeclareVariable(float3,anchor,,);
*/

rtDeclareVariable(Ray,dRay,rtCurrentRay,);
rtDeclareVariable(Intersection,aInset,attribute aInset,);

RT_PROGRAM void PgmIntersect_plane(int primIdx){
	float t=-dRay.origin.y/dRay.direction.y;
	if(t>dRay.tmin && t<dRay.tmax){
		if(rtPotentialIntersection(t)){
			aInset.geoNor=aInset.meshNor=make_float3(0.f,1.f,0.f);
			aInset.nodeIndex=-1;
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void PgmBounds_plane(int,float result[6]){
	optix::Aabb* aabb=(optix::Aabb*)result;
	aabb->m_min.x=-M_MAX_FLOAT;
	aabb->m_min.y=-0.01f;
	aabb->m_min.z=-M_MAX_FLOAT;
	aabb->m_max.x=M_MAX_FLOAT;
	aabb->m_max.y=0.0f;
	aabb->m_max.z=M_MAX_FLOAT;
}