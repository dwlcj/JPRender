#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu\optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "cu_common.h"
#include "intersection.h"
#include "../../host_device/data_instance.h"
//#include "components\intersection_refinement.h"

using namespace optix;

rtDeclareVariable(uint2,dLd,rtLaunchDim,);
rtDeclareVariable(uint2,dLi,rtLaunchIndex,);
rtDeclareVariable(Ray,dRay,rtCurrentRay,);
rtDeclareVariable(Intersection,aInset,attribute aInset,);
rtDeclareVariable(SData_Instance,V_C_DATA_INSTANCE,,);

RT_FUNC_F void GetMeshData(int bufferID,const int3& idx,float3& p0,float3& p1,float3& p2){
	p0=GET_F3_1D(bufferID,idx.x);
	p1=GET_F3_1D(bufferID,idx.y);
	p2=GET_F3_1D(bufferID,idx.z);
}

RT_FUNC_F void GetMeshData2(int bufferID,const int3& idx,float2& p0,float2& p1,float2& p2){
	p0=GET_F2_1D(bufferID,idx.x);
	p1=GET_F2_1D(bufferID,idx.y);
	p2=GET_F2_1D(bufferID,idx.z);
}

RT_PROGRAM void PgmIntersect_mesh_triangle(int primIdx){
#if 0
	//testing
	if(rtPotentialIntersection(.5f*primIdx)){
		aNodeIndex=primIdx;
		aInset.back_hit_point=make_float3(1.f);
		aInset.front_hit_point=make_float3(1.f);
		aInset.geoNor=make_float3(1.f);
		aInset.meshNor=make_float3(1.f);
		aInset.texCoord=make_float2(1.f);
		rtReportIntersection(0);
	}
	return;
#endif

	const int3 v_idx=GET_I3_1D(V_C_DATA_INSTANCE.indexBufferID,primIdx);

	Pos p0,p1,p2;
	GetMeshData(V_C_DATA_INSTANCE.vertexBufferID,v_idx,p0,p1,p2);

	// Intersect ray with triangle
	Nor n;
	float t,beta,gamma;
	if(intersect_triangle(dRay,p0,p1,p2,n,t,beta,gamma)){
		if(rtPotentialIntersection(t)){
			aInset.meshNor=normalize(n);
			float3 n0,n1,n2;
			GetMeshData(V_C_DATA_INSTANCE.norBufferID,v_idx,n0,n1,n2);
			aInset.geoNor=normalize(n1*beta+n2*gamma+n0*(1.0f-beta-gamma));

			if(V_C_DATA_INSTANCE.tanBufferSize<=0){
				aInset.tbn.Tangent=normalize(cross(aInset.geoNor,-dRay.direction));
			} else{
				float3 st0,st1,st2;
				GetMeshData(V_C_DATA_INSTANCE.tanBufferID,v_idx,st0,st1,st2);
				float3 shading_tangent=normalize(st1*beta+st2*gamma+st0*(1.0f-beta-gamma));
				aInset.tbn.Tangent=normalize(shading_tangent);
			}
			aInset.tbn.Normal=aInset.geoNor;
			aInset.tbn.Bitangent=normalize(cross(aInset.tbn.Tangent,aInset.tbn.Normal));

			float2 t0,t1,t2;
			GetMeshData2(V_C_DATA_INSTANCE.uvBufferID,v_idx,t0,t1,t2);
			aInset.texCoord=t1*beta+t2*gamma+t0*(1.0f-beta-gamma);

			aInset.nodeIndex=V_C_DATA_INSTANCE.nodeIndex;

			/*
			refine_and_offset_hitpoint(
				dRay.origin+t*dRay.direction,
				dRay.direction,
				aInset.meshNor,
				p0,
				aInset.back_hit_point,
				aInset.front_hit_point);

			aInset.back_hit_point=dRay.origin+t*dRay.direction;
			aInset.front_hit_point=dRay.origin+t*dRay.direction;
				*/
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void PgmBounds_mesh_triangle(int primIdx,float result[6]){
	Aabb* aabb=(Aabb*)result;

	rtPrintf("MeshData.indexBufferID is :%d\n",V_C_DATA_INSTANCE.vertexBufferID);
	//rtPrintf("triangle count is :%d\n",V_C_DATA_INSTANCE.triangleCount);

	const int3 v_idx=GET_I3_1D(V_C_DATA_INSTANCE.indexBufferID,primIdx);
	rtPrintf("vertex index is :%d,%d,%d\n",v_idx.x,v_idx.y,v_idx.z);

	Pos p0,p1,p2;
	GetMeshData(V_C_DATA_INSTANCE.vertexBufferID,v_idx,p0,p1,p2);
	rtPrintf("triangle first vertex pos is :%f,%f,%f\n",p0.x,p0.y,p0.z);

	const float area=length(cross(p1-p0,p2-p0));
	if(area>0.0f&&!isinf(area)){
		aabb->m_min=fminf(fminf(p0,p1),p2);
		aabb->m_max=fmaxf(fmaxf(p0,p1),p2);
	} else{
		aabb->invalidate();
	}
}

