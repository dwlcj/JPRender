#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu\optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "cu_common.h"
#include "intersection.h"

using namespace optix;

/*
rtDeclareVariable(float4,sphere,,);
*/

rtDeclareVariable(Ray,dRay,rtCurrentRay,);
rtDeclareVariable(Intersection,aInset,attribute aInset,);

RT_PROGRAM void PgmIntersect_sphere(int primIdx){
	float3 center=make_float3(0.f); //float3 center=make_float3(sphere);
	float3 O=dRay.origin-center;
	float3 D=dRay.direction;
	float radius=1.f; //float radius=sphere.w;

	float b=dot(O,D);
	float c=dot(O,O)-radius*radius;
	float disc=b*b-c;
	if(disc > 0.0f){
		float sdisc=sqrtf(disc);
		float root1=(-b-sdisc);

		float root11=0.0f;
		bool check_second=true;
		if(rtPotentialIntersection(root1+root11)){
			aInset.geoNor=(O+(root1+root11)*D)/radius;
			aInset.nodeIndex=-1;
			if(rtReportIntersection(0))
				check_second=false;
		}
		if(check_second){
			float root2=-b+sdisc;
			if(rtPotentialIntersection(root2)){
				aInset.geoNor=(O+root2*D)/radius;
				aInset.nodeIndex=-1;
				rtReportIntersection(0);
			}
		}
	}
}

RT_PROGRAM void PgmBounds_sphere(int,float result[6]){
	optix::Aabb* aabb=(optix::Aabb*)result;
	aabb->m_min.x=-1.f;
	aabb->m_min.y=-1.f;
	aabb->m_min.z=-1.f;
	aabb->m_max.x=1.f;
	aabb->m_max.y=1.0f;
	aabb->m_max.z=1.f;
}