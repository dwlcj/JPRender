#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu\optixu_matrix_namespace.h>
#include "cu_common.h"
#include "../../host_device/data_context.h"
#include "../../host_device/data_renderer.h"
#include "../../host_device/data_instance.h"
#include "intersection.h"

using namespace optix;

struct SData_prd_normalRendering{
  Nor sn_g;
  bool hitted;
};

rtDeclareVariable(Ray,dRay,rtCurrentRay,);
rtDeclareVariable(float,dT,rtIntersectionDistance,);
rtDeclareVariable(uint2,dLd,rtLaunchDim,);
rtDeclareVariable(uint2,dLi,rtLaunchIndex,);
rtDeclareVariable(SData_Context,V_C_DATA_CONTEXT,,);
rtDeclareVariable(SData_Renderer,V_C_DATA_RENDERER,,);
rtDeclareVariable(SData_Instance,V_C_DATA_INSTANCE,,);
rtDeclareVariable(rtObject,V_C_TOP_OBJECT,,);
rtDeclareVariable(SData_prd_normalRendering,prd_normalRendering,rtPayload,);
rtDeclareVariable(Intersection,aInset,attribute aInset,);

rtDeclareVariable(rtCallableProgramId<uint(uint,uint)>,CF_GETRANDOMSEED,,);
rtDeclareVariable(rtCallableProgramId<RandomFloat1(uint&)>,CF_GETSAMPLE1,,);
rtDeclareVariable(rtCallableProgramId<RandomFloat2(uint&)>,CF_GETSAMPLE2,,);
rtDeclareVariable(rtCallableProgramId<Nor(int,const Pos&,const Nor&)>,CF_GETTRIPLANENORMAL,,);
rtDeclareVariable(rtCallableProgramId<Color4(int,int,UV2)>,CF_GETTEXTURE_DIFFUSE,,);
rtDeclareVariable(rtCallableProgramId<Nor(const SData_Texture&,Intersection)>,CF_GETTEXTURE_NORMAL,,);

#define __ONLY_FUNC__
#include "common_texture_operation.h"
#include "prepare_launch.h"
#undef __ONLY_FUNC__


RT_PROGRAM void PgmLaunch_normal(){
	Nor direction; uint seed;
	PrepareLaunch(direction,seed);

  SData_prd_normalRendering prd;
  prd.hitted=false;
	prd.sn_g=make_float3(0.f);
  Ray ray=make_Ray(V_C_DATA_RENDERER.camera_pos,direction,OptixRayTypes::NORMAL,V_C_DATA_RENDERER.ray_clamp_min,V_C_DATA_RENDERER.ray_clamp_max);
  rtTrace(V_C_TOP_OBJECT,ray,prd);
  Radiance4 L;
  if(prd.hitted){

		if(V_C_DATA_RENDERER.enableShowCameraSpaceNormal){
			Nor normal_eyespace=(length(prd.sn_g)>0.f)?normalize(V_C_DATA_RENDERER.matrix_WorldtoCamera * prd.sn_g):make_float3(0.,0.,1.);
			L=make_float4(F_NORMAL_TO_COLOR(normal_eyespace),1.f);
		} else{
			L=make_float4(F_NORMAL_TO_COLOR(prd.sn_g),1.f);
		}
		if(V_C_DATA_RENDERER.frame_current>1u){
			const float weight=1.f/(float)V_C_DATA_RENDERER.frame_current;
			Radiance4 tmp=GET_F4_2D(V_C_DATA_RENDERER.output_launchBufID,dLi);
			L=lerp(tmp,L,weight);
		} 
	} else{

		if(V_C_DATA_RENDERER.enableShowCameraSpaceNormal){
			L=make_float4(0.f,0.f,1.f,1.f);
		} else{
			L=make_float4(1.f);
		}
  }
	SET_F4_2D(V_C_DATA_RENDERER.output_launchBufID,dLi,L);

}


RT_PROGRAM void PgmCH_normal(){
	Nor geoNor_g;
	const SData_Material& md=
		rtBufferId<SData_Material,1>(V_C_DATA_INSTANCE.materialBufferID)[V_C_DATA_INSTANCE.materialBufferOffset];
	switch(md.materialIndex){
	case MATERIAL_LEATHER_INDEX:
	{
		Pos hitPos_g=dRay.origin+dRay.direction*dT;
		GetNormal_triplanar(md.GE_texture,hitPos_g,geoNor_g);
	}
	break;
	default:
	{
		const SData_Texture& data=md.GE_texture;
		Nor nor=CF_GETTEXTURE_NORMAL(data,aInset);
		geoNor_g=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,nor));
	}
	break;
	}
	prd_normalRendering.sn_g=geoNor_g;
  prd_normalRendering.hitted=true;
}

