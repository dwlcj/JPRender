#include "hip/hip_runtime.h"
﻿#include <optix_device.h>
#include <optixu\optixu_math_namespace.h>
#include <optixu\optixu_matrix_namespace.h>
#include "cu_common.h"
#include "per_ray_data.h"
#include "../../host_device/data_context.h"
#include "../../host_device/data_renderer.h"

using namespace optix;

#pragma region declare

rtDeclareVariable(Ray,dRay,rtCurrentRay,);
rtDeclareVariable(uint2,dLd,rtLaunchDim,);
rtDeclareVariable(uint2,dLi,rtLaunchIndex,);
rtDeclareVariable(rtObject,V_C_TOP_OBJECT,,);
rtDeclareVariable(SData_Renderer,V_C_DATA_RENDERER,,);


rtDeclareVariable(rtCallableProgramId<uint(uint,uint)>,CF_GETRANDOMSEED,,);
rtDeclareVariable(rtCallableProgramId<RandomFloat2(uint&)>,CF_GETSAMPLE2,,);
rtDeclareVariable(rtCallableProgramId<Dir(const RandomFloat2&,float,float)>,CF_COSINESAMPLEHEMISPHERE_QUICKER,,);
rtDeclareVariable(rtCallableProgramId<float(int,int,const Nor&,float,Color4&)>,CF_GETENVRADIANCE,,);
rtDeclareVariable(rtCallableProgramId<void(RandomFloat1,RandomFloat1,RandomVariable1*,RandomVariable1*)>,CF_CONCENTRICSAMPLEDISK,,);

#pragma endregion

#define __ONLY_FUNC__
#include "common_AO_launch.h"
#include "prepare_launch.h"
#undef __ONLY_FUNC__


RT_FUNC_F float luminanceCIE(const float4& rgba){
	const float4 ntsc_luminance={0.30f, 0.59f, 0.11f,0.f};
	return dot(rgba,ntsc_luminance);
}

RT_PROGRAM void PgmLaunch_radiance(){

	/// 1. 通用计算；
	int previousMaterialIndex=MATERIAL_NONE_INDEX;
	Nor rayDirection; uint seed;
	PrepareLaunch(rayDirection,seed);
	PRD prd;


	/// 2. 计算景深
	if(V_C_DATA_RENDERER.camera_enableDOF){
		const float dirLen=length(rayDirection);
		const float wLen=length(V_C_DATA_RENDERER.camera_W);
		const float t=dirLen*V_C_DATA_RENDERER.camera_focalLength/wLen;
		rayDirection=normalize(rayDirection);
		const Pos p1=V_C_DATA_RENDERER.camera_pos+t*rayDirection;

		// lens sample;
		RandomFloat2 rf2=CF_GETSAMPLE2(seed);
		RandomVariable2 out;
		CF_CONCENTRICSAMPLEDISK(rf2.x,rf2.y,&out.x,&out.y);
		out*=V_C_DATA_RENDERER.camera_lensRadius;
		prd.hp_g=V_C_DATA_RENDERER.camera_pos+(out.x*V_C_DATA_RENDERER.camera_U+out.y*V_C_DATA_RENDERER.camera_V);

		// Ray direction;
		rayDirection=p1-prd.hp_g;
	} else{
		prd.hp_g=V_C_DATA_RENDERER.camera_pos;
	}
	rayDirection=normalize(rayDirection);


	/// 4. 迭代发射
	float4 historySum=make_float4(1.f);
	bool firstFlag=true;
	Radiance4 L_final=make_float4(0.f,0.f,0.f,1.f);
	float4 nor_final=make_float4(0.f);
	Albedo4 albedo_final=make_float4(1.f);
	prd.missed=false;
	prd.uDepth=0u;
	prd.lastMaterialEnvBlend=10.f;
	prd.seed=seed;
	prd.materialIndex=MATERIAL_NONE_INDEX;
	Ray ray=make_Ray(prd.hp_g,rayDirection,OptixRayTypes::RADIANCE,V_C_DATA_RENDERER.ray_clamp_min,V_C_DATA_RENDERER.ray_clamp_max);

	while(prd.uDepth<=V_C_DATA_RENDERER.ray_iterateDepth){
		/// 4.0 发射射线；
		rtTrace(V_C_TOP_OBJECT,ray,prd);

#if 0
		//test
		if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
			rtPrintf("wo_g[%f,%f,%f],hit[%f,%f,%f],nor[%f,%f,%f],",
				-rayDirection.x,-rayDirection.y,-rayDirection.z,
				prd.hp_g.x,prd.hp_g.y,prd.hp_g.z,
				prd.n_g.x,prd.n_g.y,prd.n_g.z
			);
			rtPrintf("wi_g[%f,%f,%f],color_out[%f,%f,%f]\n",
				prd.wi_g.x,prd.wi_g.y,prd.wi_g.z,
				prd.color_out.x,prd.color_out.y,prd.color_out.z
			);
			rtPrintf("history[%f,%f,%f],lumi[%f],BRDF[%f],PDF[%f],depth[%d],seed[%d]\n",
				historySum.x,historySum.y,historySum.z,
				prd.luminance_out,
				prd.BRDF,
				prd.PDF,
				prd.uDepth,
				prd.seed
			);
		}
#endif


		if(dot(historySum,prd.color_out)<=M_EPSILON)break;

		prd.color_out*=2.f/(1.f+luminanceCIE(prd.color_out));

		/// 4.2 累积计算结果；
		L_final+=prd.color_out*prd.luminance_out*historySum;
		if(firstFlag){
			firstFlag=false;
			if(V_C_DATA_RENDERER.enableAIdenoise){
				Nor4 normal_eyespace=make_float4((length(prd.n_g)>0.f)?normalize(V_C_DATA_RENDERER.matrix_WorldtoCamera * prd.n_g):make_float3(0.f,0.f,1.f),1.f);
				if(V_C_DATA_RENDERER.frame_current>1u){
					const float weight=1.f/(float)V_C_DATA_RENDERER.frame_current;

					float4 tmp;
					tmp=GET_F4_2D(V_C_DATA_RENDERER.output_stageNormalBufID,dLi);
					normal_eyespace=lerp(tmp,normal_eyespace,weight);

					tmp=GET_F4_2D(V_C_DATA_RENDERER.output_stageAlbedoBufID,dLi);
					prd.albedo=lerp(tmp,prd.albedo,weight);
				}
				SET_F4_2D(V_C_DATA_RENDERER.output_stageNormalBufID,dLi,normal_eyespace);
				SET_F4_2D(V_C_DATA_RENDERER.output_stageAlbedoBufID,dLi,prd.albedo);
			}
			historySum*=prd.color_out;
		} else{
			if(previousMaterialIndex==MATERIAL_GLASS_INDEX){
				historySum*=prd.color_out;
			}
		}


		if(prd.missed)break;


		/// 4.3 累乘系数，用于迭代；
		historySum*=prd.BRDF;
		historySum*=fabsf(dot(prd.n_g,prd.wi_g));
		historySum/=prd.PDF;
		if(luminanceCIE(historySum)<V_C_DATA_RENDERER.radiance_importanceThreshold)break;

		rayDirection=prd.wi_g;
		ray.origin=prd.hp_g;
		ray.direction=rayDirection;
		previousMaterialIndex=prd.materialIndex;
		++prd.uDepth;
	}// end while


	/// x. 辉度补偿；
	if(prd.uDepth>V_C_DATA_RENDERER.ray_iterateDepth){
		Color4 color;
		float L=CF_GETENVRADIANCE(V_C_DATA_RENDERER.skybox_TSID_show,V_C_DATA_RENDERER.bufferID_lights,rayDirection,prd.lastMaterialEnvBlend,color);
		L_final+=color*L*historySum;
#if 0
		if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
			rtPrintf("amended L[%f,%f,%f]\n",L*color.x,L*color.y,L*color.z);
		}
#endif
	}

#if 0
	if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
		rtPrintf("L_final[%f,%f,%f]\n",L_final.x,L_final.y,L_final.z);
	}
#endif



	/// 5. 更新辉度数据；
	const float weight=1.f/(float)V_C_DATA_RENDERER.frame_current;
	if(V_C_DATA_RENDERER.frame_current>1u){
		Radiance4 tmp=GET_F4_2D(V_C_DATA_RENDERER.output_FinalRadianceBufID,dLi);
		L_final=lerp(tmp,L_final,weight);
	}
	SET_F4_2D(V_C_DATA_RENDERER.output_FinalRadianceBufID,dLi,L_final);


	/// 6. 计算AO系数
	float AO_final=1.f;
	if(V_C_DATA_RENDERER.enableAO){
		AO_final=AOCommonLaunch(seed);

		if(V_C_DATA_RENDERER.frame_current>1u){
			float preSum=GET_F_2D(V_C_DATA_RENDERER.output_AODesityBufID,dLi);
			AO_final=lerp(preSum,AO_final,weight);
		}
		SET_F_2D(V_C_DATA_RENDERER.output_AODesityBufID,dLi,AO_final);
	}


	/// 7. 调制辉度与AO
	SET_F4_2D(V_C_DATA_RENDERER.output_launchBufID,dLi,L_final*AO_final);


#if 0
	if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
		rtPrintf("AO_final[%f]\n",AO_final);
	}
#endif

}

RT_PROGRAM void PgmException(){
	/*
	const int index=dLi.y*dLd.x+dLi.x;
	F_SET_FLOAT4_TO_BUFFER(V_C_DATA_RENDERER.output_launchBufID,index,V_C_DATA_RENDERER.color_bad);
	*/
	SET_F4_2D(V_C_DATA_RENDERER.output_launchBufID,dLi,V_C_DATA_RENDERER.color_bad);
#if 1
	//test
	//result=make_float3(GetEnvironmentTextureColor4(dRay.rayDirection));
	const uint code=rtGetExceptionCode();
	if(code!=RT_EXCEPTION_STACK_OVERFLOW){
		if(uint(dLd.x*.5f)==dLi.x && uint(dLd.y*.5f)==dLi.y){
			rtPrintExceptionDetails();
		}
		//rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n",code,dLi.x,dLi.y);
	}
#endif
}
